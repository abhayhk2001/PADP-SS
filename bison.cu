#include "hip/hip_runtime.h"
/* A Bison parser, made by GNU Bison 3.0.4.  */

/* Bison implementation for Yacc-like parsers in C

   Copyright (C) 1984, 1989-1990, 2000-2015 Free Software Foundation, Inc.

   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.  */

/* As a special exception, you may create a larger work that contains
   part or all of the Bison parser skeleton and distribute that work
   under terms of your choice, so long as that work isn't itself a
   parser generator using the skeleton or a modified version thereof
   as a parser skeleton.  Alternatively, if you modify or redistribute
   the parser skeleton itself, you may (at your option) remove this
   special exception, which will cause the skeleton and the resulting
   Bison output files to be licensed under the GNU General Public
   License without this special exception.

   This special exception was added by the Free Software Foundation in
   version 2.2 of Bison.  */

/* C LALR(1) parser skeleton written by Richard Stallman, by
   simplifying the original so-called "semantic" parser.  */

/* All symbols defined below should begin with yy or YY, to avoid
   infringing on user name space.  This should be done even for local
   variables, as they might otherwise be expanded by user macros.
   There are some unavoidable exceptions within include files to
   define necessary library symbols; they are noted "INFRINGES ON
   USER NAME SPACE" below.  */

/* Identify Bison output.  */
#define YYBISON 1

/* Bison version.  */
#define YYBISON_VERSION "3.0.4"

/* Skeleton name.  */
#define YYSKELETON_NAME "yacc.c"

/* Pure parsers.  */
#define YYPURE 0

/* Push parsers.  */
#define YYPUSH 0

/* Pull parsers.  */
#define YYPULL 1




/* Copy the first part of user declarations.  */
#line 15 "bison.y" /* yacc.c:339  */



#include "lex.yy.c"
#include "cm.h"
#include "operators.h"



#line 76 "bison.cu" /* yacc.c:339  */

# ifndef YY_NULLPTR
#  if defined __cplusplus && 201103L <= __cplusplus
#   define YY_NULLPTR nullptr
#  else
#   define YY_NULLPTR 0
#  endif
# endif

/* Enabling verbose error messages.  */
#ifdef YYERROR_VERBOSE
# undef YYERROR_VERBOSE
# define YYERROR_VERBOSE 1
#else
# define YYERROR_VERBOSE 0
#endif


/* Debug traces.  */
#ifndef YYDEBUG
# define YYDEBUG 0
#endif
#if YYDEBUG
extern int yydebug;
#endif

/* Token type.  */
#ifndef YYTOKENTYPE
# define YYTOKENTYPE
  enum yytokentype
  {
    FILENAME = 258,
    NAME = 259,
    STRING = 260,
    INTNUM = 261,
    DECIMAL1 = 262,
    BOOL1 = 263,
    APPROXNUM = 264,
    USERVAR = 265,
    ASSIGN = 266,
    EQUAL = 267,
    NONEQUAL = 268,
    OR = 269,
    XOR = 270,
    AND = 271,
    DISTINCT = 272,
    IN = 273,
    IS = 274,
    LIKE = 275,
    REGEXP = 276,
    NOT = 277,
    BETWEEN = 278,
    COMPARISON = 279,
    SHIFT = 280,
    MOD = 281,
    FROM = 282,
    DELETE = 283,
    LOAD = 284,
    FILTER = 285,
    BY = 286,
    JOIN = 287,
    STORE = 288,
    INTO = 289,
    GROUP = 290,
    SELECT = 291,
    AS = 292,
    ORDER = 293,
    ASC = 294,
    DESC = 295,
    COUNT = 296,
    USING = 297,
    SUM = 298,
    AVG = 299,
    MIN = 300,
    MAX = 301,
    LIMIT = 302,
    ON = 303,
    BINARY = 304,
    YEAR = 305,
    MONTH = 306,
    DAY = 307,
    CAST_TO_INT = 308,
    LEFT = 309,
    RIGHT = 310,
    OUTER = 311,
    SEMI = 312,
    ANTI = 313,
    SORT = 314,
    SEGMENTS = 315,
    PRESORTED = 316,
    PARTITION = 317,
    INSERT = 318,
    WHERE = 319,
    DISPLAY = 320,
    CASE = 321,
    WHEN = 322,
    THEN = 323,
    ELSE = 324,
    END = 325,
    SHOW = 326,
    TABLES = 327,
    TABLE = 328,
    DESCRIBE = 329,
    DROP = 330,
    CREATE = 331,
    INDEX = 332,
    INTERVAL = 333,
    APPEND = 334,
    NO = 335,
    ENCODING = 336
  };
#endif

/* Value type.  */
#if ! defined YYSTYPE && ! defined YYSTYPE_IS_DECLARED

union YYSTYPE
{
#line 25 "bison.y" /* yacc.c:355  */

    long long int intval;
    double floatval;
    char *strval;
    int subtok;

#line 202 "bison.cu" /* yacc.c:355  */
};

typedef union YYSTYPE YYSTYPE;
# define YYSTYPE_IS_TRIVIAL 1
# define YYSTYPE_IS_DECLARED 1
#endif


extern YYSTYPE yylval;

int yyparse (void);



/* Copy the second part of user declarations.  */

#line 219 "bison.cu" /* yacc.c:358  */

#ifdef short
# undef short
#endif

#ifdef YYTYPE_UINT8
typedef YYTYPE_UINT8 yytype_uint8;
#else
typedef unsigned char yytype_uint8;
#endif

#ifdef YYTYPE_INT8
typedef YYTYPE_INT8 yytype_int8;
#else
typedef signed char yytype_int8;
#endif

#ifdef YYTYPE_UINT16
typedef YYTYPE_UINT16 yytype_uint16;
#else
typedef unsigned short int yytype_uint16;
#endif

#ifdef YYTYPE_INT16
typedef YYTYPE_INT16 yytype_int16;
#else
typedef short int yytype_int16;
#endif

#ifndef YYSIZE_T
# ifdef __SIZE_TYPE__
#  define YYSIZE_T __SIZE_TYPE__
# elif defined size_t
#  define YYSIZE_T size_t
# elif ! defined YYSIZE_T
#  include <stddef.h> /* INFRINGES ON USER NAME SPACE */
#  define YYSIZE_T size_t
# else
#  define YYSIZE_T unsigned int
# endif
#endif

#define YYSIZE_MAXIMUM ((YYSIZE_T) -1)

#ifndef YY_
# if defined YYENABLE_NLS && YYENABLE_NLS
#  if ENABLE_NLS
#   include <libintl.h> /* INFRINGES ON USER NAME SPACE */
#   define YY_(Msgid) dgettext ("bison-runtime", Msgid)
#  endif
# endif
# ifndef YY_
#  define YY_(Msgid) Msgid
# endif
#endif

#ifndef YY_ATTRIBUTE
# if (defined __GNUC__                                               \
      && (2 < __GNUC__ || (__GNUC__ == 2 && 96 <= __GNUC_MINOR__)))  \
     || defined __SUNPRO_C && 0x5110 <= __SUNPRO_C
#  define YY_ATTRIBUTE(Spec) __attribute__(Spec)
# else
#  define YY_ATTRIBUTE(Spec) /* empty */
# endif
#endif

#ifndef YY_ATTRIBUTE_PURE
# define YY_ATTRIBUTE_PURE   YY_ATTRIBUTE ((__pure__))
#endif

#ifndef YY_ATTRIBUTE_UNUSED
# define YY_ATTRIBUTE_UNUSED YY_ATTRIBUTE ((__unused__))
#endif

#if !defined _Noreturn \
     && (!defined __STDC_VERSION__ || __STDC_VERSION__ < 201112)
# if defined _MSC_VER && 1200 <= _MSC_VER
#  define _Noreturn __declspec (noreturn)
# else
#  define _Noreturn YY_ATTRIBUTE ((__noreturn__))
# endif
#endif

/* Suppress unused-variable warnings by "using" E.  */
#if ! defined lint || defined __GNUC__
# define YYUSE(E) ((void) (E))
#else
# define YYUSE(E) /* empty */
#endif

#if defined __GNUC__ && 407 <= __GNUC__ * 100 + __GNUC_MINOR__
/* Suppress an incorrect diagnostic about yylval being uninitialized.  */
# define YY_IGNORE_MAYBE_UNINITIALIZED_BEGIN \
    _Pragma ("GCC diagnostic push") \
    _Pragma ("GCC diagnostic ignored \"-Wuninitialized\"")\
    _Pragma ("GCC diagnostic ignored \"-Wmaybe-uninitialized\"")
# define YY_IGNORE_MAYBE_UNINITIALIZED_END \
    _Pragma ("GCC diagnostic pop")
#else
# define YY_INITIAL_VALUE(Value) Value
#endif
#ifndef YY_IGNORE_MAYBE_UNINITIALIZED_BEGIN
# define YY_IGNORE_MAYBE_UNINITIALIZED_BEGIN
# define YY_IGNORE_MAYBE_UNINITIALIZED_END
#endif
#ifndef YY_INITIAL_VALUE
# define YY_INITIAL_VALUE(Value) /* Nothing. */
#endif


#if ! defined yyoverflow || YYERROR_VERBOSE

/* The parser invokes alloca or malloc; define the necessary symbols.  */

# ifdef YYSTACK_USE_ALLOCA
#  if YYSTACK_USE_ALLOCA
#   ifdef __GNUC__
#    define YYSTACK_ALLOC __builtin_alloca
#   elif defined __BUILTIN_VA_ARG_INCR
#    include <alloca.h> /* INFRINGES ON USER NAME SPACE */
#   elif defined _AIX
#    define YYSTACK_ALLOC __alloca
#   elif defined _MSC_VER
#    include <malloc.h> /* INFRINGES ON USER NAME SPACE */
#    define alloca _alloca
#   else
#    define YYSTACK_ALLOC alloca
#    if ! defined _ALLOCA_H && ! defined EXIT_SUCCESS
#     include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
      /* Use EXIT_SUCCESS as a witness for stdlib.h.  */
#     ifndef EXIT_SUCCESS
#      define EXIT_SUCCESS 0
#     endif
#    endif
#   endif
#  endif
# endif

# ifdef YYSTACK_ALLOC
   /* Pacify GCC's 'empty if-body' warning.  */
#  define YYSTACK_FREE(Ptr) do { /* empty */; } while (0)
#  ifndef YYSTACK_ALLOC_MAXIMUM
    /* The OS might guarantee only one guard page at the bottom of the stack,
       and a page size can be as small as 4096 bytes.  So we cannot safely
       invoke alloca (N) if N exceeds 4096.  Use a slightly smaller number
       to allow for a few compiler-allocated temporary stack slots.  */
#   define YYSTACK_ALLOC_MAXIMUM 4032 /* reasonable circa 2006 */
#  endif
# else
#  define YYSTACK_ALLOC YYMALLOC
#  define YYSTACK_FREE YYFREE
#  ifndef YYSTACK_ALLOC_MAXIMUM
#   define YYSTACK_ALLOC_MAXIMUM YYSIZE_MAXIMUM
#  endif
#  if (defined __cplusplus && ! defined EXIT_SUCCESS \
       && ! ((defined YYMALLOC || defined malloc) \
             && (defined YYFREE || defined free)))
#   include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#   ifndef EXIT_SUCCESS
#    define EXIT_SUCCESS 0
#   endif
#  endif
#  ifndef YYMALLOC
#   define YYMALLOC malloc
#   if ! defined malloc && ! defined EXIT_SUCCESS
void *malloc (YYSIZE_T); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
#  ifndef YYFREE
#   define YYFREE free
#   if ! defined free && ! defined EXIT_SUCCESS
void free (void *); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
# endif
#endif /* ! defined yyoverflow || YYERROR_VERBOSE */


#if (! defined yyoverflow \
     && (! defined __cplusplus \
         || (defined YYSTYPE_IS_TRIVIAL && YYSTYPE_IS_TRIVIAL)))

/* A type that is properly aligned for any stack member.  */
union yyalloc
{
  yytype_int16 yyss_alloc;
  YYSTYPE yyvs_alloc;
};

/* The size of the maximum gap between one aligned stack and the next.  */
# define YYSTACK_GAP_MAXIMUM (sizeof (union yyalloc) - 1)

/* The size of an array large to enough to hold all stacks, each with
   N elements.  */
# define YYSTACK_BYTES(N) \
     ((N) * (sizeof (yytype_int16) + sizeof (YYSTYPE)) \
      + YYSTACK_GAP_MAXIMUM)

# define YYCOPY_NEEDED 1

/* Relocate STACK from its old location to the new one.  The
   local variables YYSIZE and YYSTACKSIZE give the old and new number of
   elements in the stack, and YYPTR gives the new location of the
   stack.  Advance YYPTR to a properly aligned location for the next
   stack.  */
# define YYSTACK_RELOCATE(Stack_alloc, Stack)                           \
    do                                                                  \
      {                                                                 \
        YYSIZE_T yynewbytes;                                            \
        YYCOPY (&yyptr->Stack_alloc, Stack, yysize);                    \
        Stack = &yyptr->Stack_alloc;                                    \
        yynewbytes = yystacksize * sizeof (*Stack) + YYSTACK_GAP_MAXIMUM; \
        yyptr += yynewbytes / sizeof (*yyptr);                          \
      }                                                                 \
    while (0)

#endif

#if defined YYCOPY_NEEDED && YYCOPY_NEEDED
/* Copy COUNT objects from SRC to DST.  The source and destination do
   not overlap.  */
# ifndef YYCOPY
#  if defined __GNUC__ && 1 < __GNUC__
#   define YYCOPY(Dst, Src, Count) \
      __builtin_memcpy (Dst, Src, (Count) * sizeof (*(Src)))
#  else
#   define YYCOPY(Dst, Src, Count)              \
      do                                        \
        {                                       \
          YYSIZE_T yyi;                         \
          for (yyi = 0; yyi < (Count); yyi++)   \
            (Dst)[yyi] = (Src)[yyi];            \
        }                                       \
      while (0)
#  endif
# endif
#endif /* !YYCOPY_NEEDED */

/* YYFINAL -- State number of the termination state.  */
#define YYFINAL  23
/* YYLAST -- Last index in YYTABLE.  */
#define YYLAST   837

/* YYNTOKENS -- Number of terminals.  */
#define YYNTOKENS  99
/* YYNNTS -- Number of nonterminals.  */
#define YYNNTS  14
/* YYNRULES -- Number of rules.  */
#define YYNRULES  101
/* YYNSTATES -- Number of states.  */
#define YYNSTATES  303

/* YYTRANSLATE[YYX] -- Symbol number corresponding to YYX as returned
   by yylex, with out-of-bounds checking.  */
#define YYUNDEFTOK  2
#define YYMAXUTOK   336

#define YYTRANSLATE(YYX)                                                \
  ((unsigned int) (YYX) <= YYMAXUTOK ? yytranslate[YYX] : YYUNDEFTOK)

/* YYTRANSLATE[TOKEN-NUM] -- Symbol number corresponding to TOKEN-NUM
   as returned by yylex, without out-of-bounds checking.  */
static const yytype_uint8 yytranslate[] =
{
       0,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    23,     2,     2,     2,    33,    27,     2,
      92,    93,    31,    29,    95,    30,    94,    32,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,    98,    91,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,    35,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    96,    26,    97,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     1,     2,     3,     4,
       5,     6,     7,     8,     9,    10,    11,    12,    13,    14,
      15,    16,    17,    18,    19,    20,    21,    22,    24,    25,
      28,    34,    36,    37,    38,    39,    40,    41,    42,    43,
      44,    45,    46,    47,    48,    49,    50,    51,    52,    53,
      54,    55,    56,    57,    58,    59,    60,    61,    62,    63,
      64,    65,    66,    67,    68,    69,    70,    71,    72,    73,
      74,    75,    76,    77,    78,    79,    80,    81,    82,    83,
      84,    85,    86,    87,    88,    89,    90
};

#if YYDEBUG
  /* YYRLINE[YYN] -- Source line where rule number YYN was defined.  */
static const yytype_uint16 yyrline[] =
{
       0,   128,   128,   129,   133,   136,   138,   140,   142,   144,
     146,   148,   150,   152,   154,   156,   158,   160,   162,   164,
     166,   168,   174,   175,   176,   177,   178,   179,   180,   181,
     182,   183,   184,   185,   186,   187,   188,   189,   190,   191,
     192,   193,   194,   195,   196,   197,   198,   202,   203,   204,
     205,   206,   207,   208,   209,   210,   211,   212,   213,   214,
     215,   216,   217,   219,   220,   221,   225,   226,   229,   232,
     236,   237,   238,   242,   243,   247,   248,   251,   253,   256,
     260,   261,   262,   263,   264,   265,   266,   267,   268,   269,
     270,   271,   272,   273,   274,   275,   277,   280,   282,   285,
     286,   287
};
#endif

#if YYDEBUG || YYERROR_VERBOSE || 0
/* YYTNAME[SYMBOL-NUM] -- String name of the symbol SYMBOL-NUM.
   First, the terminals, then, starting at YYNTOKENS, nonterminals.  */
static const char *const yytname[] =
{
  "$end", "error", "$undefined", "FILENAME", "NAME", "STRING", "INTNUM",
  "DECIMAL1", "BOOL1", "APPROXNUM", "USERVAR", "ASSIGN", "EQUAL",
  "NONEQUAL", "OR", "XOR", "AND", "DISTINCT", "IN", "IS", "LIKE", "REGEXP",
  "NOT", "'!'", "BETWEEN", "COMPARISON", "'|'", "'&'", "SHIFT", "'+'",
  "'-'", "'*'", "'/'", "'%'", "MOD", "'^'", "FROM", "DELETE", "LOAD",
  "FILTER", "BY", "JOIN", "STORE", "INTO", "GROUP", "SELECT", "AS",
  "ORDER", "ASC", "DESC", "COUNT", "USING", "SUM", "AVG", "MIN", "MAX",
  "LIMIT", "ON", "BINARY", "YEAR", "MONTH", "DAY", "CAST_TO_INT", "LEFT",
  "RIGHT", "OUTER", "SEMI", "ANTI", "SORT", "SEGMENTS", "PRESORTED",
  "PARTITION", "INSERT", "WHERE", "DISPLAY", "CASE", "WHEN", "THEN",
  "ELSE", "END", "SHOW", "TABLES", "TABLE", "DESCRIBE", "DROP", "CREATE",
  "INDEX", "INTERVAL", "APPEND", "NO", "ENCODING", "';'", "'('", "')'",
  "'.'", "','", "'{'", "'}'", "':'", "$accept", "stmt_list", "stmt",
  "select_stmt", "expr", "opt_group_list", "expr_list", "load_list",
  "val_list", "opt_val_list", "opt_where", "join_list", "opt_limit",
  "sort_def", YY_NULLPTR
};
#endif

# ifdef YYPRINT
/* YYTOKNUM[NUM] -- (External) token number corresponding to the
   (internal) symbol number NUM (which must be that of a token).  */
static const yytype_uint16 yytoknum[] =
{
       0,   256,   257,   258,   259,   260,   261,   262,   263,   264,
     265,   266,   267,   268,   269,   270,   271,   272,   273,   274,
     275,   276,   277,    33,   278,   279,   124,    38,   280,    43,
      45,    42,    47,    37,   281,    94,   282,   283,   284,   285,
     286,   287,   288,   289,   290,   291,   292,   293,   294,   295,
     296,   297,   298,   299,   300,   301,   302,   303,   304,   305,
     306,   307,   308,   309,   310,   311,   312,   313,   314,   315,
     316,   317,   318,   319,   320,   321,   322,   323,   324,   325,
     326,   327,   328,   329,   330,   331,   332,   333,   334,   335,
     336,    59,    40,    41,    46,    44,   123,   125,    58
};
# endif

#define YYPACT_NINF -186

#define yypact_value_is_default(Yystate) \
  (!!((Yystate) == (-186)))

#define YYTABLE_NINF -1

#define yytable_value_is_error(Yytable_value) \
  (!!((Yytable_value) == (-1)))

  /* YYPACT[STATE-NUM] -- Index in YYTABLE of the portion describing
     STATE-NUM.  */
static const yytype_int16 yypact[] =
{
     107,    -5,   -18,    22,    -1,    34,   -36,    45,   -30,   -73,
      93,   -38,  -186,   -22,    76,    38,    87,    44,  -186,  -186,
     108,   111,   114,  -186,     7,  -186,   117,   132,   200,   133,
      69,   143,   103,    58,  -186,   104,   109,  -186,   118,   120,
      51,  -186,  -186,  -186,  -186,  -186,  -186,   259,   259,   259,
    -186,    78,    80,    82,    83,    88,    90,    91,    92,    96,
     110,   259,   703,   -31,   128,   259,   -48,   200,   182,   197,
     198,   121,   259,  -186,  -186,  -186,   207,   199,   209,   787,
     255,   255,   259,   259,   259,   259,   259,   259,   259,   259,
     259,   259,   344,   259,   259,   259,   259,   259,     2,   259,
     293,   259,   259,   259,   259,   259,   259,   259,   212,   215,
     259,   259,   749,   134,   214,   169,   170,   -25,   136,   140,
     144,   236,   749,   147,  -186,   149,   367,   390,   413,   436,
     459,   482,   505,   528,   551,   626,  -186,   749,   771,   307,
     668,   787,  -186,   239,   803,    79,   676,   164,   124,   124,
    -186,  -186,  -186,  -186,  -186,   -29,   726,    94,  -186,  -186,
     245,  -186,   191,   -27,   247,   169,   252,   253,   165,  -186,
     172,  -186,  -186,  -186,  -186,  -186,  -186,  -186,  -186,  -186,
     259,  -186,    -2,   178,   268,   233,   -39,   -37,   237,  -186,
     230,   273,   259,   186,   -27,   221,   251,  -186,  -186,  -186,
     -62,   201,   248,   289,   597,  -186,   238,   259,   300,   264,
     265,   303,   267,   276,   320,  -186,  -186,  -186,   169,  -186,
     285,   324,  -186,   325,   326,   241,   250,   259,   259,  -186,
     274,   340,   345,   304,   346,   358,   308,  -186,   362,  -186,
     277,   278,   259,   361,   574,   649,   259,   327,   331,   259,
     332,   333,   259,   322,   355,  -186,   749,   -45,    10,  -186,
    -186,   649,   259,   259,   649,   259,   259,   649,   354,   403,
    -186,   259,   319,   405,  -186,   649,   649,  -186,   649,   649,
    -186,   406,   318,   749,   341,   321,  -186,  -186,  -186,  -186,
    -186,   412,  -186,  -186,   357,   430,   323,   431,   424,   435,
     359,   450,  -186
};

  /* YYDEFACT[STATE-NUM] -- Default reduction number in state STATE-NUM.
     Performed when YYTABLE does not specify something else to do.  Zero
     means the default is an error.  */
static const yytype_uint8 yydefact[] =
{
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     4,     0,     0,     0,     0,     0,    17,    13,
       0,     0,     0,     1,     0,     2,     0,     0,     0,     0,
       0,     0,     0,     0,    18,     0,     0,     3,     0,     0,
      22,    25,    26,    27,    29,    28,    24,     0,     0,     0,
      72,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,    96,     0,     0,     0,
       0,     0,     0,     7,    34,    35,     0,     0,     0,    41,
      59,    60,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,    77,    15,     0,     0,    96,     0,     0,     0,     0,
       0,     0,    79,     0,    23,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,    64,    54,    55,    56,
      57,    53,    66,     0,    62,     0,    61,    58,    47,    48,
      49,    50,    51,    52,    70,    68,     0,    75,    78,     8,
       0,    97,     0,    98,     0,    96,     0,     0,     0,    46,
       0,    36,    37,    38,    39,    40,    42,    43,    44,    45,
       0,    67,    22,     0,     0,     0,     0,     0,     0,     5,
      68,     0,     0,     0,    98,     0,     0,    11,    14,    16,
       0,     0,     0,     0,     0,    63,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     9,    71,    76,    96,    12,
       0,     0,    20,     0,     0,     0,    33,     0,     0,    69,
       0,     0,     0,     0,     0,     0,     0,    10,     0,   101,
       0,     0,     0,     0,     0,    80,     0,     0,     0,     0,
       0,     0,     0,    99,     0,    21,    73,     0,     0,    65,
      88,    84,     0,     0,    85,     0,     0,    87,     0,     0,
       6,     0,    32,     0,    91,    83,    81,    93,    86,    82,
      95,     0,     0,    74,     0,     0,    92,    89,    94,    90,
     100,     0,    31,    30,     0,     0,     0,     0,     0,     0,
       0,     0,    19
};

  /* YYPGOTO[NTERM-NUM].  */
static const yytype_int16 yypgoto[] =
{
    -186,  -186,   447,   295,   -28,   269,   391,  -186,  -185,  -186,
    -186,   -34,  -114,   282
};

  /* YYDEFGOTO[NTERM-NUM].  */
static const yytype_int16 yydefgoto[] =
{
      -1,    10,    11,    12,   157,   189,    63,   257,   158,   159,
      73,   190,   116,   197
};

  /* YYTABLE[YYPACT[STATE-NUM]] -- What to do in state STATE-NUM.  If
     positive, shift that token.  If negative, reduce the rule whose
     number is the opposite.  If YYTABLE_NINF, syntax error.  */
static const yytype_int16 yytable[] =
{
      62,   162,   208,   113,   211,   109,    13,   217,   114,    13,
     142,   164,   184,    21,    22,   185,    26,    27,    14,    79,
      80,    81,   229,    28,   143,    29,    15,   209,   210,   212,
     213,   222,   223,    92,   186,   187,   188,   112,    17,    62,
     115,   195,    16,   196,   122,    18,    74,    75,   270,    19,
     271,   199,    20,    25,   126,   127,   128,   129,   130,   131,
     132,   133,   134,   135,   110,   137,   138,   139,   140,   141,
     110,   144,   146,   147,   148,   149,   150,   151,   152,   153,
      30,    31,   156,   182,    41,    42,    43,    44,    45,    46,
      76,    32,    77,    23,    78,    33,    47,     1,    37,    74,
      75,    48,    49,   272,   237,   273,    93,    94,    95,    96,
      97,     1,    34,    98,    99,    35,     2,    92,    36,   100,
      38,     3,   101,   102,   103,   104,   105,   106,   107,    51,
       2,    52,    53,    54,    55,     3,    39,    64,    56,    57,
      58,    59,    65,    76,     2,    77,    66,    78,    67,     3,
      68,     4,   204,     5,    60,   104,   105,   106,   107,     6,
      72,    69,     7,     8,     9,     4,    70,     5,   111,    71,
      82,    61,    83,     6,    84,    85,     7,     8,     9,     4,
      86,     5,    87,    88,    89,   118,    91,     6,    90,   192,
       7,     8,     9,   102,   103,   104,   105,   106,   107,   244,
     245,   119,   120,   124,    40,    41,    42,    43,    44,    45,
      46,   260,   123,   121,   256,   125,   154,    47,   261,   155,
     161,   264,    48,    49,   267,   114,   160,   274,   163,   165,
     277,    50,   166,   280,   275,   276,   167,   278,   279,   168,
     169,   286,   287,   283,   288,   289,   170,   181,   193,   194,
      51,   198,    52,    53,    54,    55,   200,   201,   202,    56,
      57,    58,    59,    40,    41,    42,    43,    44,    45,    46,
     203,   205,   206,   207,   185,    60,    47,   216,   214,   218,
     100,    48,    49,   101,   102,   103,   104,   105,   106,   107,
     220,   221,    61,   226,   225,   228,   224,    40,    41,    42,
      43,    44,    45,    46,   230,   231,   232,   233,   234,    51,
      47,    52,    53,    54,    55,    48,    49,   235,    56,    57,
      58,    59,    96,    97,   236,   238,    98,    99,   239,   240,
     241,   246,   100,   242,    60,   101,   102,   103,   104,   105,
     106,   107,   243,    51,   247,    52,    53,    54,    55,   248,
     250,    61,    56,    57,    58,    59,    93,    94,    95,    96,
      97,   249,   251,    98,    99,   252,   253,   258,    60,   100,
     254,   255,   101,   102,   103,   104,   105,   106,   107,    93,
      94,    95,    96,    97,   262,   145,    98,    99,   263,   265,
     266,   269,   100,   268,   281,   101,   102,   103,   104,   105,
     106,   107,    93,    94,    95,    96,    97,   282,   284,    98,
      99,   285,   290,   291,   293,   100,   294,   297,   101,   102,
     103,   104,   105,   106,   107,    93,    94,    95,    96,    97,
     295,   292,    98,    99,   296,   298,   299,   136,   100,   300,
     183,   101,   102,   103,   104,   105,   106,   107,    93,    94,
      95,    96,    97,   301,   302,    98,    99,    24,   117,   215,
     171,   100,     0,     0,   101,   102,   103,   104,   105,   106,
     107,    93,    94,    95,    96,    97,   219,     0,    98,    99,
       0,     0,     0,   172,   100,     0,     0,   101,   102,   103,
     104,   105,   106,   107,    93,    94,    95,    96,    97,     0,
       0,    98,    99,     0,     0,     0,   173,   100,     0,     0,
     101,   102,   103,   104,   105,   106,   107,    93,    94,    95,
      96,    97,     0,     0,    98,    99,     0,     0,     0,   174,
     100,     0,     0,   101,   102,   103,   104,   105,   106,   107,
      93,    94,    95,    96,    97,     0,     0,    98,    99,     0,
       0,     0,   175,   100,     0,     0,   101,   102,   103,   104,
     105,   106,   107,    93,    94,    95,    96,    97,     0,     0,
      98,    99,     0,     0,     0,   176,   100,     0,     0,   101,
     102,   103,   104,   105,   106,   107,    93,    94,    95,    96,
      97,     0,     0,    98,    99,     0,     0,     0,   177,   100,
       0,     0,   101,   102,   103,   104,   105,   106,   107,    93,
      94,    95,    96,    97,     0,     0,    98,    99,     0,     0,
       0,   178,   100,     0,     0,   101,   102,   103,   104,   105,
     106,   107,     0,     0,     0,     0,     0,     0,    93,    94,
      95,    96,    97,     0,   179,    98,    99,     0,     0,     0,
       0,   100,     0,   259,   101,   102,   103,   104,   105,   106,
     107,    93,    94,    95,    96,    97,     0,     0,    98,    99,
       0,     0,     0,     0,   100,   227,     0,   101,   102,   103,
     104,   105,   106,   107,    97,     0,     0,    98,    99,     0,
     184,     0,     0,   100,     0,     0,   101,   102,   103,   104,
     105,   106,   107,   180,   101,   102,   103,   104,   105,   106,
     107,     0,   186,   187,   188,    93,    94,    95,    96,    97,
       0,     0,    98,    99,     0,     0,     0,     0,   100,     0,
       0,   101,   102,   103,   104,   105,   106,   107,    93,    94,
      95,    96,    97,     0,     0,    98,    99,     0,     0,   108,
       0,   100,     0,     0,   101,   102,   103,   104,   105,   106,
     107,    93,    94,    95,    96,    97,     0,     0,    98,    99,
       0,     0,   191,     0,   100,     0,     0,   101,   102,   103,
     104,   105,   106,   107,    94,    95,    96,    97,     0,     0,
      98,    99,     0,     0,     0,     0,   100,     0,     0,   101,
     102,   103,   104,   105,   106,   107,    98,    99,     0,     0,
       0,     0,   100,     0,     0,   101,   102,   103,   104,   105,
     106,   107,    -1,    -1,     0,     0,     0,     0,   100,     0,
       0,   101,   102,   103,   104,   105,   106,   107
};

static const yytype_int16 yycheck[] =
{
      28,   115,    41,    51,    41,    36,    11,   192,    56,    11,
       8,    36,    41,    86,    87,    44,    38,    39,    36,    47,
      48,    49,   207,    45,    22,    47,     4,    66,    67,    66,
      67,    93,    94,    61,    63,    64,    65,    65,     4,    67,
      88,    68,    43,    70,    72,    81,    48,    49,    93,     4,
      95,   165,    82,    91,    82,    83,    84,    85,    86,    87,
      88,    89,    90,    91,    95,    93,    94,    95,    96,    97,
      95,    99,   100,   101,   102,   103,   104,   105,   106,   107,
       4,    43,   110,     4,     5,     6,     7,     8,     9,    10,
      92,     4,    94,     0,    96,    51,    17,     4,    91,    48,
      49,    22,    23,    93,   218,    95,    12,    13,    14,    15,
      16,     4,     4,    19,    20,     4,    37,   145,     4,    25,
       3,    42,    28,    29,    30,    31,    32,    33,    34,    50,
      37,    52,    53,    54,    55,    42,     4,     4,    59,    60,
      61,    62,    73,    92,    37,    94,     3,    96,    45,    42,
      92,    72,   180,    74,    75,    31,    32,    33,    34,    80,
      40,    57,    83,    84,    85,    72,    57,    74,    40,    51,
      92,    92,    92,    80,    92,    92,    83,    84,    85,    72,
      92,    74,    92,    92,    92,     3,    76,    80,    92,    95,
      83,    84,    85,    29,    30,    31,    32,    33,    34,   227,
     228,     4,     4,     4,     4,     5,     6,     7,     8,     9,
      10,   245,     5,    92,   242,     6,     4,    17,   246,     4,
       6,   249,    22,    23,   252,    56,    92,   261,    58,    93,
     264,    31,    92,   267,   262,   263,    92,   265,   266,     3,
      93,   275,   276,   271,   278,   279,    97,     8,     3,    58,
      50,     4,    52,    53,    54,    55,     4,     4,    93,    59,
      60,    61,    62,     4,     5,     6,     7,     8,     9,    10,
      98,    93,     4,    40,    44,    75,    17,     4,    41,    93,
      25,    22,    23,    28,    29,    30,    31,    32,    33,    34,
      69,    40,    92,     4,    46,    57,    95,     4,     5,     6,
       7,     8,     9,    10,     4,    41,    41,     4,    41,    50,
      17,    52,    53,    54,    55,    22,    23,    41,    59,    60,
      61,    62,    15,    16,     4,    40,    19,    20,     4,     4,
       4,    57,    25,    92,    75,    28,    29,    30,    31,    32,
      33,    34,    92,    50,     4,    52,    53,    54,    55,     4,
       4,    92,    59,    60,    61,    62,    12,    13,    14,    15,
      16,    57,     4,    19,    20,    57,     4,     6,    75,    25,
      93,    93,    28,    29,    30,    31,    32,    33,    34,    12,
      13,    14,    15,    16,    57,    92,    19,    20,    57,    57,
      57,    36,    25,    71,    40,    28,    29,    30,    31,    32,
      33,    34,    12,    13,    14,    15,    16,     4,    89,    19,
      20,     6,     6,    95,    93,    25,     4,    94,    28,    29,
      30,    31,    32,    33,    34,    12,    13,    14,    15,    16,
      73,    90,    19,    20,     4,     4,    12,    93,    25,     4,
     145,    28,    29,    30,    31,    32,    33,    34,    12,    13,
      14,    15,    16,    94,     4,    19,    20,    10,    67,   190,
      93,    25,    -1,    -1,    28,    29,    30,    31,    32,    33,
      34,    12,    13,    14,    15,    16,   194,    -1,    19,    20,
      -1,    -1,    -1,    93,    25,    -1,    -1,    28,    29,    30,
      31,    32,    33,    34,    12,    13,    14,    15,    16,    -1,
      -1,    19,    20,    -1,    -1,    -1,    93,    25,    -1,    -1,
      28,    29,    30,    31,    32,    33,    34,    12,    13,    14,
      15,    16,    -1,    -1,    19,    20,    -1,    -1,    -1,    93,
      25,    -1,    -1,    28,    29,    30,    31,    32,    33,    34,
      12,    13,    14,    15,    16,    -1,    -1,    19,    20,    -1,
      -1,    -1,    93,    25,    -1,    -1,    28,    29,    30,    31,
      32,    33,    34,    12,    13,    14,    15,    16,    -1,    -1,
      19,    20,    -1,    -1,    -1,    93,    25,    -1,    -1,    28,
      29,    30,    31,    32,    33,    34,    12,    13,    14,    15,
      16,    -1,    -1,    19,    20,    -1,    -1,    -1,    93,    25,
      -1,    -1,    28,    29,    30,    31,    32,    33,    34,    12,
      13,    14,    15,    16,    -1,    -1,    19,    20,    -1,    -1,
      -1,    93,    25,    -1,    -1,    28,    29,    30,    31,    32,
      33,    34,    -1,    -1,    -1,    -1,    -1,    -1,    12,    13,
      14,    15,    16,    -1,    93,    19,    20,    -1,    -1,    -1,
      -1,    25,    -1,    79,    28,    29,    30,    31,    32,    33,
      34,    12,    13,    14,    15,    16,    -1,    -1,    19,    20,
      -1,    -1,    -1,    -1,    25,    78,    -1,    28,    29,    30,
      31,    32,    33,    34,    16,    -1,    -1,    19,    20,    -1,
      41,    -1,    -1,    25,    -1,    -1,    28,    29,    30,    31,
      32,    33,    34,    77,    28,    29,    30,    31,    32,    33,
      34,    -1,    63,    64,    65,    12,    13,    14,    15,    16,
      -1,    -1,    19,    20,    -1,    -1,    -1,    -1,    25,    -1,
      -1,    28,    29,    30,    31,    32,    33,    34,    12,    13,
      14,    15,    16,    -1,    -1,    19,    20,    -1,    -1,    46,
      -1,    25,    -1,    -1,    28,    29,    30,    31,    32,    33,
      34,    12,    13,    14,    15,    16,    -1,    -1,    19,    20,
      -1,    -1,    46,    -1,    25,    -1,    -1,    28,    29,    30,
      31,    32,    33,    34,    13,    14,    15,    16,    -1,    -1,
      19,    20,    -1,    -1,    -1,    -1,    25,    -1,    -1,    28,
      29,    30,    31,    32,    33,    34,    19,    20,    -1,    -1,
      -1,    -1,    25,    -1,    -1,    28,    29,    30,    31,    32,
      33,    34,    19,    20,    -1,    -1,    -1,    -1,    25,    -1,
      -1,    28,    29,    30,    31,    32,    33,    34
};

  /* YYSTOS[STATE-NUM] -- The (internal number of the) accessing
     symbol of state STATE-NUM.  */
static const yytype_uint8 yystos[] =
{
       0,     4,    37,    42,    72,    74,    80,    83,    84,    85,
     100,   101,   102,    11,    36,     4,    43,     4,    81,     4,
      82,    86,    87,     0,   101,    91,    38,    39,    45,    47,
       4,    43,     4,    51,     4,     4,     4,    91,     3,     4,
       4,     5,     6,     7,     8,     9,    10,    17,    22,    23,
      31,    50,    52,    53,    54,    55,    59,    60,    61,    62,
      75,    92,   103,   105,     4,    73,     3,    45,    92,    57,
      57,    51,    40,   109,    48,    49,    92,    94,    96,   103,
     103,   103,    92,    92,    92,    92,    92,    92,    92,    92,
      92,    76,   103,    12,    13,    14,    15,    16,    19,    20,
      25,    28,    29,    30,    31,    32,    33,    34,    46,    36,
      95,    40,   103,    51,    56,    88,   111,   105,     3,     4,
       4,    92,   103,     5,     4,     6,   103,   103,   103,   103,
     103,   103,   103,   103,   103,   103,    93,   103,   103,   103,
     103,   103,     8,    22,   103,    92,   103,   103,   103,   103,
     103,   103,   103,   103,     4,     4,   103,   103,   107,   108,
      92,     6,   111,    58,    36,    93,    92,    92,     3,    93,
      97,    93,    93,    93,    93,    93,    93,    93,    93,    93,
      77,     8,     4,   102,    41,    44,    63,    64,    65,   104,
     110,    46,    95,     3,    58,    68,    70,   112,     4,   111,
       4,     4,    93,    98,   103,    93,     4,    40,    41,    66,
      67,    41,    66,    67,    41,   104,     4,   107,    93,   112,
      69,    40,    93,    94,    95,    46,     4,    78,    57,   107,
       4,    41,    41,     4,    41,    41,     4,   111,    40,     4,
       4,     4,    92,    92,   103,   103,    57,     4,     4,    57,
       4,     4,    57,     4,    93,    93,   103,   106,     6,    79,
     110,   103,    57,    57,   103,    57,    57,   103,    71,    36,
      93,    95,    93,    95,   110,   103,   103,   110,   103,   103,
     110,    40,     4,   103,    89,     6,   110,   110,   110,   110,
       6,    95,    90,    93,     4,    73,     4,    94,     4,    12,
       4,    94,     4
};

  /* YYR1[YYN] -- Symbol number of symbol that rule YYN derives.  */
static const yytype_uint8 yyr1[] =
{
       0,    99,   100,   100,   101,   102,   102,   102,   102,   102,
     102,   102,   102,   102,   102,   102,   102,   102,   102,   102,
     102,   102,   103,   103,   103,   103,   103,   103,   103,   103,
     103,   103,   103,   103,   103,   103,   103,   103,   103,   103,
     103,   103,   103,   103,   103,   103,   103,   103,   103,   103,
     103,   103,   103,   103,   103,   103,   103,   103,   103,   103,
     103,   103,   103,   103,   103,   103,   103,   103,   104,   104,
     105,   105,   105,   106,   106,   107,   107,   108,   108,   109,
     110,   110,   110,   110,   110,   110,   110,   110,   110,   110,
     110,   110,   110,   110,   110,   110,   111,   111,   112,   112,
     112,   112
};

  /* YYR2[YYN] -- Number of symbols on the right hand side of rule YYN.  */
static const yytype_uint8 yyr2[] =
{
       0,     2,     2,     3,     1,     7,    12,     5,     6,     8,
       9,     7,     8,     2,     7,     5,     7,     2,     3,    22,
       8,    10,     1,     3,     1,     1,     1,     1,     1,     1,
      11,    11,     9,     6,     2,     2,     4,     4,     4,     4,
       4,     2,     4,     4,     4,     4,     4,     3,     3,     3,
       3,     3,     3,     3,     3,     3,     3,     3,     3,     2,
       2,     3,     3,     5,     3,     8,     3,     4,     0,     3,
       3,     5,     1,     1,     3,     1,     3,     0,     1,     2,
       4,     6,     6,     6,     5,     5,     6,     5,     5,     7,
       7,     6,     7,     6,     7,     6,     0,     2,     0,     4,
       7,     3
};


#define yyerrok         (yyerrstatus = 0)
#define yyclearin       (yychar = YYEMPTY)
#define YYEMPTY         (-2)
#define YYEOF           0

#define YYACCEPT        goto yyacceptlab
#define YYABORT         goto yyabortlab
#define YYERROR         goto yyerrorlab


#define YYRECOVERING()  (!!yyerrstatus)

#define YYBACKUP(Token, Value)                                  \
do                                                              \
  if (yychar == YYEMPTY)                                        \
    {                                                           \
      yychar = (Token);                                         \
      yylval = (Value);                                         \
      YYPOPSTACK (yylen);                                       \
      yystate = *yyssp;                                         \
      goto yybackup;                                            \
    }                                                           \
  else                                                          \
    {                                                           \
      yyerror (YY_("syntax error: cannot back up")); \
      YYERROR;                                                  \
    }                                                           \
while (0)

/* Error token number */
#define YYTERROR        1
#define YYERRCODE       256



/* Enable debugging if requested.  */
#if YYDEBUG

# ifndef YYFPRINTF
#  include <stdio.h> /* INFRINGES ON USER NAME SPACE */
#  define YYFPRINTF fprintf
# endif

# define YYDPRINTF(Args)                        \
do {                                            \
  if (yydebug)                                  \
    YYFPRINTF Args;                             \
} while (0)

/* This macro is provided for backward compatibility. */
#ifndef YY_LOCATION_PRINT
# define YY_LOCATION_PRINT(File, Loc) ((void) 0)
#endif


# define YY_SYMBOL_PRINT(Title, Type, Value, Location)                    \
do {                                                                      \
  if (yydebug)                                                            \
    {                                                                     \
      YYFPRINTF (stderr, "%s ", Title);                                   \
      yy_symbol_print (stderr,                                            \
                  Type, Value); \
      YYFPRINTF (stderr, "\n");                                           \
    }                                                                     \
} while (0)


/*----------------------------------------.
| Print this symbol's value on YYOUTPUT.  |
`----------------------------------------*/

static void
yy_symbol_value_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
{
  FILE *yyo = yyoutput;
  YYUSE (yyo);
  if (!yyvaluep)
    return;
# ifdef YYPRINT
  if (yytype < YYNTOKENS)
    YYPRINT (yyoutput, yytoknum[yytype], *yyvaluep);
# endif
  YYUSE (yytype);
}


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

static void
yy_symbol_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
{
  YYFPRINTF (yyoutput, "%s %s (",
             yytype < YYNTOKENS ? "token" : "nterm", yytname[yytype]);

  yy_symbol_value_print (yyoutput, yytype, yyvaluep);
  YYFPRINTF (yyoutput, ")");
}

/*------------------------------------------------------------------.
| yy_stack_print -- Print the state stack from its BOTTOM up to its |
| TOP (included).                                                   |
`------------------------------------------------------------------*/

static void
yy_stack_print (yytype_int16 *yybottom, yytype_int16 *yytop)
{
  YYFPRINTF (stderr, "Stack now");
  for (; yybottom <= yytop; yybottom++)
    {
      int yybot = *yybottom;
      YYFPRINTF (stderr, " %d", yybot);
    }
  YYFPRINTF (stderr, "\n");
}

# define YY_STACK_PRINT(Bottom, Top)                            \
do {                                                            \
  if (yydebug)                                                  \
    yy_stack_print ((Bottom), (Top));                           \
} while (0)


/*------------------------------------------------.
| Report that the YYRULE is going to be reduced.  |
`------------------------------------------------*/

static void
yy_reduce_print (yytype_int16 *yyssp, YYSTYPE *yyvsp, int yyrule)
{
  unsigned long int yylno = yyrline[yyrule];
  int yynrhs = yyr2[yyrule];
  int yyi;
  YYFPRINTF (stderr, "Reducing stack by rule %d (line %lu):\n",
             yyrule - 1, yylno);
  /* The symbols being reduced.  */
  for (yyi = 0; yyi < yynrhs; yyi++)
    {
      YYFPRINTF (stderr, "   $%d = ", yyi + 1);
      yy_symbol_print (stderr,
                       yystos[yyssp[yyi + 1 - yynrhs]],
                       &(yyvsp[(yyi + 1) - (yynrhs)])
                                              );
      YYFPRINTF (stderr, "\n");
    }
}

# define YY_REDUCE_PRINT(Rule)          \
do {                                    \
  if (yydebug)                          \
    yy_reduce_print (yyssp, yyvsp, Rule); \
} while (0)

/* Nonzero means print parse trace.  It is left uninitialized so that
   multiple parsers can coexist.  */
int yydebug;
#else /* !YYDEBUG */
# define YYDPRINTF(Args)
# define YY_SYMBOL_PRINT(Title, Type, Value, Location)
# define YY_STACK_PRINT(Bottom, Top)
# define YY_REDUCE_PRINT(Rule)
#endif /* !YYDEBUG */


/* YYINITDEPTH -- initial size of the parser's stacks.  */
#ifndef YYINITDEPTH
# define YYINITDEPTH 200
#endif

/* YYMAXDEPTH -- maximum size the stacks can grow to (effective only
   if the built-in stack extension method is used).

   Do not make this value too large; the results are undefined if
   YYSTACK_ALLOC_MAXIMUM < YYSTACK_BYTES (YYMAXDEPTH)
   evaluated with infinite-precision integer arithmetic.  */

#ifndef YYMAXDEPTH
# define YYMAXDEPTH 10000
#endif


#if YYERROR_VERBOSE

# ifndef yystrlen
#  if defined __GLIBC__ && defined _STRING_H
#   define yystrlen strlen
#  else
/* Return the length of YYSTR.  */
static YYSIZE_T
yystrlen (const char *yystr)
{
  YYSIZE_T yylen;
  for (yylen = 0; yystr[yylen]; yylen++)
    continue;
  return yylen;
}
#  endif
# endif

# ifndef yystpcpy
#  if defined __GLIBC__ && defined _STRING_H && defined _GNU_SOURCE
#   define yystpcpy stpcpy
#  else
/* Copy YYSRC to YYDEST, returning the address of the terminating '\0' in
   YYDEST.  */
static char *
yystpcpy (char *yydest, const char *yysrc)
{
  char *yyd = yydest;
  const char *yys = yysrc;

  while ((*yyd++ = *yys++) != '\0')
    continue;

  return yyd - 1;
}
#  endif
# endif

# ifndef yytnamerr
/* Copy to YYRES the contents of YYSTR after stripping away unnecessary
   quotes and backslashes, so that it's suitable for yyerror.  The
   heuristic is that double-quoting is unnecessary unless the string
   contains an apostrophe, a comma, or backslash (other than
   backslash-backslash).  YYSTR is taken from yytname.  If YYRES is
   null, do not copy; instead, return the length of what the result
   would have been.  */
static YYSIZE_T
yytnamerr (char *yyres, const char *yystr)
{
  if (*yystr == '"')
    {
      YYSIZE_T yyn = 0;
      char const *yyp = yystr;

      for (;;)
        switch (*++yyp)
          {
          case '\'':
          case ',':
            goto do_not_strip_quotes;

          case '\\':
            if (*++yyp != '\\')
              goto do_not_strip_quotes;
            /* Fall through.  */
          default:
            if (yyres)
              yyres[yyn] = *yyp;
            yyn++;
            break;

          case '"':
            if (yyres)
              yyres[yyn] = '\0';
            return yyn;
          }
    do_not_strip_quotes: ;
    }

  if (! yyres)
    return yystrlen (yystr);

  return yystpcpy (yyres, yystr) - yyres;
}
# endif

/* Copy into *YYMSG, which is of size *YYMSG_ALLOC, an error message
   about the unexpected token YYTOKEN for the state stack whose top is
   YYSSP.

   Return 0 if *YYMSG was successfully written.  Return 1 if *YYMSG is
   not large enough to hold the message.  In that case, also set
   *YYMSG_ALLOC to the required number of bytes.  Return 2 if the
   required number of bytes is too large to store.  */
static int
yysyntax_error (YYSIZE_T *yymsg_alloc, char **yymsg,
                yytype_int16 *yyssp, int yytoken)
{
  YYSIZE_T yysize0 = yytnamerr (YY_NULLPTR, yytname[yytoken]);
  YYSIZE_T yysize = yysize0;
  enum { YYERROR_VERBOSE_ARGS_MAXIMUM = 5 };
  /* Internationalized format string. */
  const char *yyformat = YY_NULLPTR;
  /* Arguments of yyformat. */
  char const *yyarg[YYERROR_VERBOSE_ARGS_MAXIMUM];
  /* Number of reported tokens (one for the "unexpected", one per
     "expected"). */
  int yycount = 0;

  /* There are many possibilities here to consider:
     - If this state is a consistent state with a default action, then
       the only way this function was invoked is if the default action
       is an error action.  In that case, don't check for expected
       tokens because there are none.
     - The only way there can be no lookahead present (in yychar) is if
       this state is a consistent state with a default action.  Thus,
       detecting the absence of a lookahead is sufficient to determine
       that there is no unexpected or expected token to report.  In that
       case, just report a simple "syntax error".
     - Don't assume there isn't a lookahead just because this state is a
       consistent state with a default action.  There might have been a
       previous inconsistent state, consistent state with a non-default
       action, or user semantic action that manipulated yychar.
     - Of course, the expected token list depends on states to have
       correct lookahead information, and it depends on the parser not
       to perform extra reductions after fetching a lookahead from the
       scanner and before detecting a syntax error.  Thus, state merging
       (from LALR or IELR) and default reductions corrupt the expected
       token list.  However, the list is correct for canonical LR with
       one exception: it will still contain any token that will not be
       accepted due to an error action in a later state.
  */
  if (yytoken != YYEMPTY)
    {
      int yyn = yypact[*yyssp];
      yyarg[yycount++] = yytname[yytoken];
      if (!yypact_value_is_default (yyn))
        {
          /* Start YYX at -YYN if negative to avoid negative indexes in
             YYCHECK.  In other words, skip the first -YYN actions for
             this state because they are default actions.  */
          int yyxbegin = yyn < 0 ? -yyn : 0;
          /* Stay within bounds of both yycheck and yytname.  */
          int yychecklim = YYLAST - yyn + 1;
          int yyxend = yychecklim < YYNTOKENS ? yychecklim : YYNTOKENS;
          int yyx;

          for (yyx = yyxbegin; yyx < yyxend; ++yyx)
            if (yycheck[yyx + yyn] == yyx && yyx != YYTERROR
                && !yytable_value_is_error (yytable[yyx + yyn]))
              {
                if (yycount == YYERROR_VERBOSE_ARGS_MAXIMUM)
                  {
                    yycount = 1;
                    yysize = yysize0;
                    break;
                  }
                yyarg[yycount++] = yytname[yyx];
                {
                  YYSIZE_T yysize1 = yysize + yytnamerr (YY_NULLPTR, yytname[yyx]);
                  if (! (yysize <= yysize1
                         && yysize1 <= YYSTACK_ALLOC_MAXIMUM))
                    return 2;
                  yysize = yysize1;
                }
              }
        }
    }

  switch (yycount)
    {
# define YYCASE_(N, S)                      \
      case N:                               \
        yyformat = S;                       \
      break
      YYCASE_(0, YY_("syntax error"));
      YYCASE_(1, YY_("syntax error, unexpected %s"));
      YYCASE_(2, YY_("syntax error, unexpected %s, expecting %s"));
      YYCASE_(3, YY_("syntax error, unexpected %s, expecting %s or %s"));
      YYCASE_(4, YY_("syntax error, unexpected %s, expecting %s or %s or %s"));
      YYCASE_(5, YY_("syntax error, unexpected %s, expecting %s or %s or %s or %s"));
# undef YYCASE_
    }

  {
    YYSIZE_T yysize1 = yysize + yystrlen (yyformat);
    if (! (yysize <= yysize1 && yysize1 <= YYSTACK_ALLOC_MAXIMUM))
      return 2;
    yysize = yysize1;
  }

  if (*yymsg_alloc < yysize)
    {
      *yymsg_alloc = 2 * yysize;
      if (! (yysize <= *yymsg_alloc
             && *yymsg_alloc <= YYSTACK_ALLOC_MAXIMUM))
        *yymsg_alloc = YYSTACK_ALLOC_MAXIMUM;
      return 1;
    }

  /* Avoid sprintf, as that infringes on the user's name space.
     Don't have undefined behavior even if the translation
     produced a string with the wrong number of "%s"s.  */
  {
    char *yyp = *yymsg;
    int yyi = 0;
    while ((*yyp = *yyformat) != '\0')
      if (*yyp == '%' && yyformat[1] == 's' && yyi < yycount)
        {
          yyp += yytnamerr (yyp, yyarg[yyi++]);
          yyformat += 2;
        }
      else
        {
          yyp++;
          yyformat++;
        }
  }
  return 0;
}
#endif /* YYERROR_VERBOSE */

/*-----------------------------------------------.
| Release the memory associated to this symbol.  |
`-----------------------------------------------*/

static void
yydestruct (const char *yymsg, int yytype, YYSTYPE *yyvaluep)
{
  YYUSE (yyvaluep);
  if (!yymsg)
    yymsg = "Deleting";
  YY_SYMBOL_PRINT (yymsg, yytype, yyvaluep, yylocationp);

  YY_IGNORE_MAYBE_UNINITIALIZED_BEGIN
  YYUSE (yytype);
  YY_IGNORE_MAYBE_UNINITIALIZED_END
}




/* The lookahead symbol.  */
int yychar;

/* The semantic value of the lookahead symbol.  */
YYSTYPE yylval;
/* Number of syntax errors so far.  */
int yynerrs;


/*----------.
| yyparse.  |
`----------*/

int
yyparse (void)
{
    int yystate;
    /* Number of tokens to shift before error messages enabled.  */
    int yyerrstatus;

    /* The stacks and their tools:
       'yyss': related to states.
       'yyvs': related to semantic values.

       Refer to the stacks through separate pointers, to allow yyoverflow
       to reallocate them elsewhere.  */

    /* The state stack.  */
    yytype_int16 yyssa[YYINITDEPTH];
    yytype_int16 *yyss;
    yytype_int16 *yyssp;

    /* The semantic value stack.  */
    YYSTYPE yyvsa[YYINITDEPTH];
    YYSTYPE *yyvs;
    YYSTYPE *yyvsp;

    YYSIZE_T yystacksize;

  int yyn;
  int yyresult;
  /* Lookahead token as an internal (translated) token number.  */
  int yytoken = 0;
  /* The variables used to return semantic value and location from the
     action routines.  */
  YYSTYPE yyval;

#if YYERROR_VERBOSE
  /* Buffer for error messages, and its allocated size.  */
  char yymsgbuf[128];
  char *yymsg = yymsgbuf;
  YYSIZE_T yymsg_alloc = sizeof yymsgbuf;
#endif

#define YYPOPSTACK(N)   (yyvsp -= (N), yyssp -= (N))

  /* The number of symbols on the RHS of the reduced rule.
     Keep to zero when no symbol should be popped.  */
  int yylen = 0;

  yyssp = yyss = yyssa;
  yyvsp = yyvs = yyvsa;
  yystacksize = YYINITDEPTH;

  YYDPRINTF ((stderr, "Starting parse\n"));

  yystate = 0;
  yyerrstatus = 0;
  yynerrs = 0;
  yychar = YYEMPTY; /* Cause a token to be read.  */
  goto yysetstate;

/*------------------------------------------------------------.
| yynewstate -- Push a new state, which is found in yystate.  |
`------------------------------------------------------------*/
 yynewstate:
  /* In all cases, when you get here, the value and location stacks
     have just been pushed.  So pushing a state here evens the stacks.  */
  yyssp++;

 yysetstate:
  *yyssp = yystate;

  if (yyss + yystacksize - 1 <= yyssp)
    {
      /* Get the current used size of the three stacks, in elements.  */
      YYSIZE_T yysize = yyssp - yyss + 1;

#ifdef yyoverflow
      {
        /* Give user a chance to reallocate the stack.  Use copies of
           these so that the &'s don't force the real ones into
           memory.  */
        YYSTYPE *yyvs1 = yyvs;
        yytype_int16 *yyss1 = yyss;

        /* Each stack pointer address is followed by the size of the
           data in use in that stack, in bytes.  This used to be a
           conditional around just the two extra args, but that might
           be undefined if yyoverflow is a macro.  */
        yyoverflow (YY_("memory exhausted"),
                    &yyss1, yysize * sizeof (*yyssp),
                    &yyvs1, yysize * sizeof (*yyvsp),
                    &yystacksize);

        yyss = yyss1;
        yyvs = yyvs1;
      }
#else /* no yyoverflow */
# ifndef YYSTACK_RELOCATE
      goto yyexhaustedlab;
# else
      /* Extend the stack our own way.  */
      if (YYMAXDEPTH <= yystacksize)
        goto yyexhaustedlab;
      yystacksize *= 2;
      if (YYMAXDEPTH < yystacksize)
        yystacksize = YYMAXDEPTH;

      {
        yytype_int16 *yyss1 = yyss;
        union yyalloc *yyptr =
          (union yyalloc *) YYSTACK_ALLOC (YYSTACK_BYTES (yystacksize));
        if (! yyptr)
          goto yyexhaustedlab;
        YYSTACK_RELOCATE (yyss_alloc, yyss);
        YYSTACK_RELOCATE (yyvs_alloc, yyvs);
#  undef YYSTACK_RELOCATE
        if (yyss1 != yyssa)
          YYSTACK_FREE (yyss1);
      }
# endif
#endif /* no yyoverflow */

      yyssp = yyss + yysize - 1;
      yyvsp = yyvs + yysize - 1;

      YYDPRINTF ((stderr, "Stack size increased to %lu\n",
                  (unsigned long int) yystacksize));

      if (yyss + yystacksize - 1 <= yyssp)
        YYABORT;
    }

  YYDPRINTF ((stderr, "Entering state %d\n", yystate));

  if (yystate == YYFINAL)
    YYACCEPT;

  goto yybackup;

/*-----------.
| yybackup.  |
`-----------*/
yybackup:

  /* Do appropriate processing given the current state.  Read a
     lookahead token if we need one and don't already have one.  */

  /* First try to decide what to do without reference to lookahead token.  */
  yyn = yypact[yystate];
  if (yypact_value_is_default (yyn))
    goto yydefault;

  /* Not known => get a lookahead token if don't already have one.  */

  /* YYCHAR is either YYEMPTY or YYEOF or a valid lookahead symbol.  */
  if (yychar == YYEMPTY)
    {
      YYDPRINTF ((stderr, "Reading a token: "));
      yychar = yylex ();
    }

  if (yychar <= YYEOF)
    {
      yychar = yytoken = YYEOF;
      YYDPRINTF ((stderr, "Now at end of input.\n"));
    }
  else
    {
      yytoken = YYTRANSLATE (yychar);
      YY_SYMBOL_PRINT ("Next token is", yytoken, &yylval, &yylloc);
    }

  /* If the proper action on seeing token YYTOKEN is to reduce or to
     detect an error, take that action.  */
  yyn += yytoken;
  if (yyn < 0 || YYLAST < yyn || yycheck[yyn] != yytoken)
    goto yydefault;
  yyn = yytable[yyn];
  if (yyn <= 0)
    {
      if (yytable_value_is_error (yyn))
        goto yyerrlab;
      yyn = -yyn;
      goto yyreduce;
    }

  /* Count tokens shifted since error; after three, turn off error
     status.  */
  if (yyerrstatus)
    yyerrstatus--;

  /* Shift the lookahead token.  */
  YY_SYMBOL_PRINT ("Shifting", yytoken, &yylval, &yylloc);

  /* Discard the shifted token.  */
  yychar = YYEMPTY;

  yystate = yyn;
  YY_IGNORE_MAYBE_UNINITIALIZED_BEGIN
  *++yyvsp = yylval;
  YY_IGNORE_MAYBE_UNINITIALIZED_END

  goto yynewstate;


/*-----------------------------------------------------------.
| yydefault -- do the default action for the current state.  |
`-----------------------------------------------------------*/
yydefault:
  yyn = yydefact[yystate];
  if (yyn == 0)
    goto yyerrlab;
  goto yyreduce;


/*-----------------------------.
| yyreduce -- Do a reduction.  |
`-----------------------------*/
yyreduce:
  /* yyn is the number of a rule to reduce with.  */
  yylen = yyr2[yyn];

  /* If YYLEN is nonzero, implement the default value of the action:
     '$$ = $1'.

     Otherwise, the following line sets YYVAL to garbage.
     This behavior is undocumented and Bison
     users should not rely upon it.  Assigning to YYVAL
     unconditionally makes the parser a bit smaller, and it avoids a
     GCC warning that YYVAL may be used uninitialized.  */
  yyval = yyvsp[1-yylen];


  YY_REDUCE_PRINT (yyn);
  switch (yyn)
    {
        case 4:
#line 133 "bison.y" /* yacc.c:1646  */
    { emit("STMT"); }
#line 1603 "bison.cu" /* yacc.c:1646  */
    break;

  case 5:
#line 137 "bison.y" /* yacc.c:1646  */
    { emit_select((yyvsp[-6].strval), (yyvsp[-1].strval), (yyvsp[0].intval)); }
#line 1609 "bison.cu" /* yacc.c:1646  */
    break;

  case 6:
#line 139 "bison.y" /* yacc.c:1646  */
    {  emit_load((yyvsp[-11].strval), (yyvsp[-8].strval), (yyvsp[-1].intval), (yyvsp[-5].strval)); }
#line 1615 "bison.cu" /* yacc.c:1646  */
    break;

  case 7:
#line 141 "bison.y" /* yacc.c:1646  */
    {  emit_filter((yyvsp[-4].strval), (yyvsp[-1].strval));}
#line 1621 "bison.cu" /* yacc.c:1646  */
    break;

  case 8:
#line 143 "bison.y" /* yacc.c:1646  */
    {  emit_order((yyvsp[-5].strval), (yyvsp[-2].strval), (yyvsp[0].intval));}
#line 1627 "bison.cu" /* yacc.c:1646  */
    break;

  case 9:
#line 145 "bison.y" /* yacc.c:1646  */
    {  emit_join((yyvsp[-7].strval),(yyvsp[-2].strval),(yyvsp[-1].intval),0,-1); }
#line 1633 "bison.cu" /* yacc.c:1646  */
    break;

  case 10:
#line 147 "bison.y" /* yacc.c:1646  */
    {  emit_store((yyvsp[-7].strval),(yyvsp[-5].strval),(yyvsp[-2].strval)); }
#line 1639 "bison.cu" /* yacc.c:1646  */
    break;

  case 11:
#line 149 "bison.y" /* yacc.c:1646  */
    {  emit_store_binary((yyvsp[-5].strval),(yyvsp[-3].strval),0); }
#line 1645 "bison.cu" /* yacc.c:1646  */
    break;

  case 12:
#line 151 "bison.y" /* yacc.c:1646  */
    {  emit_store_binary((yyvsp[-6].strval),(yyvsp[-4].strval),1); }
#line 1651 "bison.cu" /* yacc.c:1646  */
    break;

  case 13:
#line 153 "bison.y" /* yacc.c:1646  */
    {  emit_describe_table((yyvsp[0].strval));}
#line 1657 "bison.cu" /* yacc.c:1646  */
    break;

  case 14:
#line 155 "bison.y" /* yacc.c:1646  */
    {  emit_insert((yyvsp[-4].strval), (yyvsp[0].strval));}
#line 1663 "bison.cu" /* yacc.c:1646  */
    break;

  case 15:
#line 157 "bison.y" /* yacc.c:1646  */
    {  emit_delete((yyvsp[-2].strval));}
#line 1669 "bison.cu" /* yacc.c:1646  */
    break;

  case 16:
#line 159 "bison.y" /* yacc.c:1646  */
    {  emit_display((yyvsp[-5].strval), (yyvsp[-2].strval));}
#line 1675 "bison.cu" /* yacc.c:1646  */
    break;

  case 17:
#line 161 "bison.y" /* yacc.c:1646  */
    {  emit_show_tables();}
#line 1681 "bison.cu" /* yacc.c:1646  */
    break;

  case 18:
#line 163 "bison.y" /* yacc.c:1646  */
    {  emit_drop_table((yyvsp[0].strval));}
#line 1687 "bison.cu" /* yacc.c:1646  */
    break;

  case 19:
#line 165 "bison.y" /* yacc.c:1646  */
    {  emit_create_bitmap_index((yyvsp[-19].strval), (yyvsp[-17].strval), (yyvsp[-15].strval), (yyvsp[-13].strval), (yyvsp[-4].strval), (yyvsp[0].strval));}
#line 1693 "bison.cu" /* yacc.c:1646  */
    break;

  case 20:
#line 167 "bison.y" /* yacc.c:1646  */
    {  emit_create_index((yyvsp[-5].strval), (yyvsp[-3].strval), (yyvsp[-1].strval));}
#line 1699 "bison.cu" /* yacc.c:1646  */
    break;

  case 21:
#line 169 "bison.y" /* yacc.c:1646  */
    {  emit_create_interval((yyvsp[-7].strval), (yyvsp[-5].strval), (yyvsp[-3].strval), (yyvsp[-1].strval));}
#line 1705 "bison.cu" /* yacc.c:1646  */
    break;

  case 22:
#line 174 "bison.y" /* yacc.c:1646  */
    { emit_name((yyvsp[0].strval)); }
#line 1711 "bison.cu" /* yacc.c:1646  */
    break;

  case 23:
#line 175 "bison.y" /* yacc.c:1646  */
    { emit_fieldname((yyvsp[-2].strval), (yyvsp[0].strval)); }
#line 1717 "bison.cu" /* yacc.c:1646  */
    break;

  case 24:
#line 176 "bison.y" /* yacc.c:1646  */
    { emit("USERVAR %s", (yyvsp[0].strval)); }
#line 1723 "bison.cu" /* yacc.c:1646  */
    break;

  case 25:
#line 177 "bison.y" /* yacc.c:1646  */
    { emit_string((yyvsp[0].strval)); }
#line 1729 "bison.cu" /* yacc.c:1646  */
    break;

  case 26:
#line 178 "bison.y" /* yacc.c:1646  */
    { emit_number((yyvsp[0].intval)); }
#line 1735 "bison.cu" /* yacc.c:1646  */
    break;

  case 27:
#line 179 "bison.y" /* yacc.c:1646  */
    { emit_decimal((yyvsp[0].strval)); }
#line 1741 "bison.cu" /* yacc.c:1646  */
    break;

  case 28:
#line 180 "bison.y" /* yacc.c:1646  */
    { emit_float((yyvsp[0].floatval)); }
#line 1747 "bison.cu" /* yacc.c:1646  */
    break;

  case 29:
#line 181 "bison.y" /* yacc.c:1646  */
    { emit("BOOL %d", (yyvsp[0].intval)); }
#line 1753 "bison.cu" /* yacc.c:1646  */
    break;

  case 30:
#line 182 "bison.y" /* yacc.c:1646  */
    { emit_vardecimal((yyvsp[-10].strval), (yyvsp[-8].intval), (yyvsp[-5].strval),  (yyvsp[-3].intval), (yyvsp[-1].intval));}
#line 1759 "bison.cu" /* yacc.c:1646  */
    break;

  case 31:
#line 183 "bison.y" /* yacc.c:1646  */
    { emit_varchar((yyvsp[-10].strval), (yyvsp[-8].intval), (yyvsp[-5].strval), (yyvsp[-3].intval), "", "", "N");}
#line 1765 "bison.cu" /* yacc.c:1646  */
    break;

  case 32:
#line 184 "bison.y" /* yacc.c:1646  */
    { emit_varchar((yyvsp[-8].strval), (yyvsp[-6].intval), (yyvsp[-3].strval), (yyvsp[-1].intval), "", "", "");}
#line 1771 "bison.cu" /* yacc.c:1646  */
    break;

  case 33:
#line 185 "bison.y" /* yacc.c:1646  */
    { emit_var((yyvsp[-5].strval), (yyvsp[-3].intval), (yyvsp[0].strval), "", "");}
#line 1777 "bison.cu" /* yacc.c:1646  */
    break;

  case 34:
#line 186 "bison.y" /* yacc.c:1646  */
    { emit_var_asc((yyvsp[-1].strval));}
#line 1783 "bison.cu" /* yacc.c:1646  */
    break;

  case 35:
#line 187 "bison.y" /* yacc.c:1646  */
    { emit_var_desc((yyvsp[-1].strval));}
#line 1789 "bison.cu" /* yacc.c:1646  */
    break;

  case 36:
#line 188 "bison.y" /* yacc.c:1646  */
    { emit_count(); }
#line 1795 "bison.cu" /* yacc.c:1646  */
    break;

  case 37:
#line 189 "bison.y" /* yacc.c:1646  */
    { emit_sum(); }
#line 1801 "bison.cu" /* yacc.c:1646  */
    break;

  case 38:
#line 190 "bison.y" /* yacc.c:1646  */
    { emit_average(); }
#line 1807 "bison.cu" /* yacc.c:1646  */
    break;

  case 39:
#line 191 "bison.y" /* yacc.c:1646  */
    { emit_min(); }
#line 1813 "bison.cu" /* yacc.c:1646  */
    break;

  case 40:
#line 192 "bison.y" /* yacc.c:1646  */
    { emit_max(); }
#line 1819 "bison.cu" /* yacc.c:1646  */
    break;

  case 41:
#line 193 "bison.y" /* yacc.c:1646  */
    { emit_distinct(); }
#line 1825 "bison.cu" /* yacc.c:1646  */
    break;

  case 42:
#line 194 "bison.y" /* yacc.c:1646  */
    { emit_year(); }
#line 1831 "bison.cu" /* yacc.c:1646  */
    break;

  case 43:
#line 195 "bison.y" /* yacc.c:1646  */
    { emit_month(); }
#line 1837 "bison.cu" /* yacc.c:1646  */
    break;

  case 44:
#line 196 "bison.y" /* yacc.c:1646  */
    { emit_day(); }
#line 1843 "bison.cu" /* yacc.c:1646  */
    break;

  case 45:
#line 197 "bison.y" /* yacc.c:1646  */
    { emit_cast(); }
#line 1849 "bison.cu" /* yacc.c:1646  */
    break;

  case 46:
#line 198 "bison.y" /* yacc.c:1646  */
    { emit_string_grp((yyvsp[-3].strval), (yyvsp[-1].strval)); }
#line 1855 "bison.cu" /* yacc.c:1646  */
    break;

  case 47:
#line 202 "bison.y" /* yacc.c:1646  */
    { emit_add(); }
#line 1861 "bison.cu" /* yacc.c:1646  */
    break;

  case 48:
#line 203 "bison.y" /* yacc.c:1646  */
    { emit_minus(); }
#line 1867 "bison.cu" /* yacc.c:1646  */
    break;

  case 49:
#line 204 "bison.y" /* yacc.c:1646  */
    { emit_mul(); }
#line 1873 "bison.cu" /* yacc.c:1646  */
    break;

  case 50:
#line 205 "bison.y" /* yacc.c:1646  */
    { emit_div(); }
#line 1879 "bison.cu" /* yacc.c:1646  */
    break;

  case 51:
#line 206 "bison.y" /* yacc.c:1646  */
    { emit("MOD"); }
#line 1885 "bison.cu" /* yacc.c:1646  */
    break;

  case 52:
#line 207 "bison.y" /* yacc.c:1646  */
    { emit("MOD"); }
#line 1891 "bison.cu" /* yacc.c:1646  */
    break;

  case 53:
#line 208 "bison.y" /* yacc.c:1646  */
    { emit_and(); }
#line 1897 "bison.cu" /* yacc.c:1646  */
    break;

  case 54:
#line 209 "bison.y" /* yacc.c:1646  */
    { emit_eq(); }
#line 1903 "bison.cu" /* yacc.c:1646  */
    break;

  case 55:
#line 210 "bison.y" /* yacc.c:1646  */
    { emit_neq(); }
#line 1909 "bison.cu" /* yacc.c:1646  */
    break;

  case 56:
#line 211 "bison.y" /* yacc.c:1646  */
    { emit_or(); }
#line 1915 "bison.cu" /* yacc.c:1646  */
    break;

  case 57:
#line 212 "bison.y" /* yacc.c:1646  */
    { emit("XOR"); }
#line 1921 "bison.cu" /* yacc.c:1646  */
    break;

  case 58:
#line 213 "bison.y" /* yacc.c:1646  */
    { emit("SHIFT %s", (yyvsp[-1].subtok)==1?"left":"right"); }
#line 1927 "bison.cu" /* yacc.c:1646  */
    break;

  case 59:
#line 214 "bison.y" /* yacc.c:1646  */
    { emit("NOT"); }
#line 1933 "bison.cu" /* yacc.c:1646  */
    break;

  case 60:
#line 215 "bison.y" /* yacc.c:1646  */
    { emit("NOT"); }
#line 1939 "bison.cu" /* yacc.c:1646  */
    break;

  case 61:
#line 216 "bison.y" /* yacc.c:1646  */
    { emit_cmp((yyvsp[-1].subtok)); }
#line 1945 "bison.cu" /* yacc.c:1646  */
    break;

  case 62:
#line 217 "bison.y" /* yacc.c:1646  */
    { emit_cmp(7); }
#line 1951 "bison.cu" /* yacc.c:1646  */
    break;

  case 63:
#line 219 "bison.y" /* yacc.c:1646  */
    { emit("CMPSELECT %d", (yyvsp[-3].subtok)); }
#line 1957 "bison.cu" /* yacc.c:1646  */
    break;

  case 64:
#line 220 "bison.y" /* yacc.c:1646  */
    {emit("EXPR");}
#line 1963 "bison.cu" /* yacc.c:1646  */
    break;

  case 65:
#line 221 "bison.y" /* yacc.c:1646  */
    { emit_case(); }
#line 1969 "bison.cu" /* yacc.c:1646  */
    break;

  case 66:
#line 225 "bison.y" /* yacc.c:1646  */
    { emit("ISBOOL %d", (yyvsp[0].intval)); }
#line 1975 "bison.cu" /* yacc.c:1646  */
    break;

  case 67:
#line 226 "bison.y" /* yacc.c:1646  */
    { emit("ISBOOL %d", (yyvsp[0].intval)); emit("NOT"); }
#line 1981 "bison.cu" /* yacc.c:1646  */
    break;

  case 68:
#line 229 "bison.y" /* yacc.c:1646  */
    { /* nil */
    (yyval.intval) = 0;
}
#line 1989 "bison.cu" /* yacc.c:1646  */
    break;

  case 69:
#line 232 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = (yyvsp[0].intval);}
#line 1995 "bison.cu" /* yacc.c:1646  */
    break;

  case 70:
#line 236 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_sel_name((yyvsp[0].strval));}
#line 2001 "bison.cu" /* yacc.c:1646  */
    break;

  case 71:
#line 237 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = (yyvsp[-4].intval) + 1; emit_sel_name((yyvsp[0].strval));}
#line 2007 "bison.cu" /* yacc.c:1646  */
    break;

  case 72:
#line 238 "bison.y" /* yacc.c:1646  */
    { emit_sel_name("*");}
#line 2013 "bison.cu" /* yacc.c:1646  */
    break;

  case 73:
#line 242 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; }
#line 2019 "bison.cu" /* yacc.c:1646  */
    break;

  case 74:
#line 243 "bison.y" /* yacc.c:1646  */
    {(yyval.intval) = (yyvsp[-2].intval) + 1; }
#line 2025 "bison.cu" /* yacc.c:1646  */
    break;

  case 75:
#line 247 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; }
#line 2031 "bison.cu" /* yacc.c:1646  */
    break;

  case 76:
#line 248 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1 + (yyvsp[0].intval); }
#line 2037 "bison.cu" /* yacc.c:1646  */
    break;

  case 77:
#line 251 "bison.y" /* yacc.c:1646  */
    { /* nil */
    (yyval.intval) = 0;
}
#line 2045 "bison.cu" /* yacc.c:1646  */
    break;

  case 79:
#line 256 "bison.y" /* yacc.c:1646  */
    { emit("FILTER BY"); }
#line 2051 "bison.cu" /* yacc.c:1646  */
    break;

  case 80:
#line 260 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-2].strval), 'I');}
#line 2057 "bison.cu" /* yacc.c:1646  */
    break;

  case 81:
#line 261 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-2].strval), '3');}
#line 2063 "bison.cu" /* yacc.c:1646  */
    break;

  case 82:
#line 262 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-2].strval), '4');}
#line 2069 "bison.cu" /* yacc.c:1646  */
    break;

  case 83:
#line 263 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-2].strval), '1');}
#line 2075 "bison.cu" /* yacc.c:1646  */
    break;

  case 84:
#line 264 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-2].strval), 'S');}
#line 2081 "bison.cu" /* yacc.c:1646  */
    break;

  case 85:
#line 265 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-2].strval), 'R');}
#line 2087 "bison.cu" /* yacc.c:1646  */
    break;

  case 86:
#line 266 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-2].strval), '2');}
#line 2093 "bison.cu" /* yacc.c:1646  */
    break;

  case 87:
#line 267 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-2].strval), 'O');}
#line 2099 "bison.cu" /* yacc.c:1646  */
    break;

  case 88:
#line 268 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-3].strval), 'I'); }
#line 2105 "bison.cu" /* yacc.c:1646  */
    break;

  case 89:
#line 269 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-3].strval), '3'); }
#line 2111 "bison.cu" /* yacc.c:1646  */
    break;

  case 90:
#line 270 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-3].strval), '4'); }
#line 2117 "bison.cu" /* yacc.c:1646  */
    break;

  case 91:
#line 271 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-3].strval), 'L'); }
#line 2123 "bison.cu" /* yacc.c:1646  */
    break;

  case 92:
#line 272 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-3].strval), '1'); }
#line 2129 "bison.cu" /* yacc.c:1646  */
    break;

  case 93:
#line 273 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-3].strval), 'R'); }
#line 2135 "bison.cu" /* yacc.c:1646  */
    break;

  case 94:
#line 274 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-3].strval), 'R'); }
#line 2141 "bison.cu" /* yacc.c:1646  */
    break;

  case 95:
#line 275 "bison.y" /* yacc.c:1646  */
    { (yyval.intval) = 1; emit_join_tab((yyvsp[-3].strval), 'O'); }
#line 2147 "bison.cu" /* yacc.c:1646  */
    break;

  case 96:
#line 277 "bison.y" /* yacc.c:1646  */
    { /* nil */
    (yyval.intval) = 0;
}
#line 2155 "bison.cu" /* yacc.c:1646  */
    break;

  case 97:
#line 280 "bison.y" /* yacc.c:1646  */
    { emit_limit((yyvsp[0].intval)); }
#line 2161 "bison.cu" /* yacc.c:1646  */
    break;

  case 98:
#line 282 "bison.y" /* yacc.c:1646  */
    { /* nil */
    (yyval.intval) = 0;
}
#line 2169 "bison.cu" /* yacc.c:1646  */
    break;

  case 99:
#line 285 "bison.y" /* yacc.c:1646  */
    { emit_sort((yyvsp[0].strval), 0); }
#line 2175 "bison.cu" /* yacc.c:1646  */
    break;

  case 100:
#line 286 "bison.y" /* yacc.c:1646  */
    { emit_sort((yyvsp[-3].strval), (yyvsp[0].intval)); }
#line 2181 "bison.cu" /* yacc.c:1646  */
    break;

  case 101:
#line 287 "bison.y" /* yacc.c:1646  */
    { emit_presort((yyvsp[0].strval)); }
#line 2187 "bison.cu" /* yacc.c:1646  */
    break;


#line 2191 "bison.cu" /* yacc.c:1646  */
      default: break;
    }
  /* User semantic actions sometimes alter yychar, and that requires
     that yytoken be updated with the new translation.  We take the
     approach of translating immediately before every use of yytoken.
     One alternative is translating here after every semantic action,
     but that translation would be missed if the semantic action invokes
     YYABORT, YYACCEPT, or YYERROR immediately after altering yychar or
     if it invokes YYBACKUP.  In the case of YYABORT or YYACCEPT, an
     incorrect destructor might then be invoked immediately.  In the
     case of YYERROR or YYBACKUP, subsequent parser actions might lead
     to an incorrect destructor call or verbose syntax error message
     before the lookahead is translated.  */
  YY_SYMBOL_PRINT ("-> $$ =", yyr1[yyn], &yyval, &yyloc);

  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);

  *++yyvsp = yyval;

  /* Now 'shift' the result of the reduction.  Determine what state
     that goes to, based on the state we popped back to and the rule
     number reduced by.  */

  yyn = yyr1[yyn];

  yystate = yypgoto[yyn - YYNTOKENS] + *yyssp;
  if (0 <= yystate && yystate <= YYLAST && yycheck[yystate] == *yyssp)
    yystate = yytable[yystate];
  else
    yystate = yydefgoto[yyn - YYNTOKENS];

  goto yynewstate;


/*--------------------------------------.
| yyerrlab -- here on detecting error.  |
`--------------------------------------*/
yyerrlab:
  /* Make sure we have latest lookahead translation.  See comments at
     user semantic actions for why this is necessary.  */
  yytoken = yychar == YYEMPTY ? YYEMPTY : YYTRANSLATE (yychar);

  /* If not already recovering from an error, report this error.  */
  if (!yyerrstatus)
    {
      ++yynerrs;
#if ! YYERROR_VERBOSE
      yyerror (YY_("syntax error"));
#else
# define YYSYNTAX_ERROR yysyntax_error (&yymsg_alloc, &yymsg, \
                                        yyssp, yytoken)
      {
        char const *yymsgp = YY_("syntax error");
        int yysyntax_error_status;
        yysyntax_error_status = YYSYNTAX_ERROR;
        if (yysyntax_error_status == 0)
          yymsgp = yymsg;
        else if (yysyntax_error_status == 1)
          {
            if (yymsg != yymsgbuf)
              YYSTACK_FREE (yymsg);
            yymsg = (char *) YYSTACK_ALLOC (yymsg_alloc);
            if (!yymsg)
              {
                yymsg = yymsgbuf;
                yymsg_alloc = sizeof yymsgbuf;
                yysyntax_error_status = 2;
              }
            else
              {
                yysyntax_error_status = YYSYNTAX_ERROR;
                yymsgp = yymsg;
              }
          }
        yyerror (yymsgp);
        if (yysyntax_error_status == 2)
          goto yyexhaustedlab;
      }
# undef YYSYNTAX_ERROR
#endif
    }



  if (yyerrstatus == 3)
    {
      /* If just tried and failed to reuse lookahead token after an
         error, discard it.  */

      if (yychar <= YYEOF)
        {
          /* Return failure if at end of input.  */
          if (yychar == YYEOF)
            YYABORT;
        }
      else
        {
          yydestruct ("Error: discarding",
                      yytoken, &yylval);
          yychar = YYEMPTY;
        }
    }

  /* Else will try to reuse lookahead token after shifting the error
     token.  */
  goto yyerrlab1;


/*---------------------------------------------------.
| yyerrorlab -- error raised explicitly by YYERROR.  |
`---------------------------------------------------*/
yyerrorlab:

  /* Pacify compilers like GCC when the user code never invokes
     YYERROR and the label yyerrorlab therefore never appears in user
     code.  */
  if (/*CONSTCOND*/ 0)
     goto yyerrorlab;

  /* Do not reclaim the symbols of the rule whose action triggered
     this YYERROR.  */
  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);
  yystate = *yyssp;
  goto yyerrlab1;


/*-------------------------------------------------------------.
| yyerrlab1 -- common code for both syntax error and YYERROR.  |
`-------------------------------------------------------------*/
yyerrlab1:
  yyerrstatus = 3;      /* Each real token shifted decrements this.  */

  for (;;)
    {
      yyn = yypact[yystate];
      if (!yypact_value_is_default (yyn))
        {
          yyn += YYTERROR;
          if (0 <= yyn && yyn <= YYLAST && yycheck[yyn] == YYTERROR)
            {
              yyn = yytable[yyn];
              if (0 < yyn)
                break;
            }
        }

      /* Pop the current state because it cannot handle the error token.  */
      if (yyssp == yyss)
        YYABORT;


      yydestruct ("Error: popping",
                  yystos[yystate], yyvsp);
      YYPOPSTACK (1);
      yystate = *yyssp;
      YY_STACK_PRINT (yyss, yyssp);
    }

  YY_IGNORE_MAYBE_UNINITIALIZED_BEGIN
  *++yyvsp = yylval;
  YY_IGNORE_MAYBE_UNINITIALIZED_END


  /* Shift the error token.  */
  YY_SYMBOL_PRINT ("Shifting", yystos[yyn], yyvsp, yylsp);

  yystate = yyn;
  goto yynewstate;


/*-------------------------------------.
| yyacceptlab -- YYACCEPT comes here.  |
`-------------------------------------*/
yyacceptlab:
  yyresult = 0;
  goto yyreturn;

/*-----------------------------------.
| yyabortlab -- YYABORT comes here.  |
`-----------------------------------*/
yyabortlab:
  yyresult = 1;
  goto yyreturn;

#if !defined yyoverflow || YYERROR_VERBOSE
/*-------------------------------------------------.
| yyexhaustedlab -- memory exhaustion comes here.  |
`-------------------------------------------------*/
yyexhaustedlab:
  yyerror (YY_("memory exhausted"));
  yyresult = 2;
  /* Fall through.  */
#endif

yyreturn:
  if (yychar != YYEMPTY)
    {
      /* Make sure we have latest lookahead translation.  See comments at
         user semantic actions for why this is necessary.  */
      yytoken = YYTRANSLATE (yychar);
      yydestruct ("Cleanup: discarding lookahead",
                  yytoken, &yylval);
    }
  /* Do not reclaim the symbols of the rule whose action triggered
     this YYABORT or YYACCEPT.  */
  YYPOPSTACK (yylen);
  YY_STACK_PRINT (yyss, yyssp);
  while (yyssp != yyss)
    {
      yydestruct ("Cleanup: popping",
                  yystos[*yyssp], yyvsp);
      YYPOPSTACK (1);
    }
#ifndef yyoverflow
  if (yyss != yyssa)
    YYSTACK_FREE (yyss);
#endif
#if YYERROR_VERBOSE
  if (yymsg != yymsgbuf)
    YYSTACK_FREE (yymsg);
#endif
  return yyresult;
}
#line 289 "bison.y" /* yacc.c:1906  */


bool scan_state;
unsigned int statement_count;
time_t curr_time;

int execute_file(int ac, char **av)
{
    bool just_once  = 0;
    string script;
    process_count = 1000000000; //1GB by default
    verbose = 0;
	ssd = 0;
	delta = 0;
    total_buffer_size = 0;
	hash_seed = 100;

    for (int i = 1; i < ac; i++) {
        if(strcmp(av[i],"-l") == 0) {
            process_count = 1000000*atoff(av[i+1]);
        }
        else if(strcmp(av[i],"-v") == 0) {
            verbose = 1;
        }
        else if(strcmp(av[i],"-delta") == 0) {
            delta = 1;
        }		
        else if(strcmp(av[i],"-ssd") == 0) {
            ssd = 1;
        }		
        else if(strcmp(av[i],"-precision") == 0) {
            prs = atoi(av[i+1]);
        }				
        else if(strcmp(av[i],"-i") == 0) {
            interactive = 1;
            break;
        }
        else if(strcmp(av[i],"-s") == 0) {
            just_once = 1;
            interactive = 1;
            script = av[i+1];
        };
    };
	
    load_col_data(data_dict, "data.dictionary");
	tot_disk = 0;

    if (!interactive) {
        if((yyin = fopen(av[ac-1], "r")) == nullptr) {
            perror(av[ac-1]);
            exit(1);
        };		
        if(yyparse()) {
            printf("SQL scan parse failed\n");
            exit(1);
        };
		
        scan_state = 1;
        std::clock_t start1 = std::clock();

        load_vars();
		
        statement_count = 0;
        clean_queues();
		filter_var.clear();

        yyin = fopen(av[ac-1], "r");
        PROC_FLUSH_BUF ( yyin );
        statement_count = 0;
        extern FILE *yyin;		
		curr_time = time(0)*1000;
        if(!yyparse()) {
            if(verbose)
                cout << "SQL scan parse worked " << endl;
        }
        else
            cout << "SQL scan parse failed" << endl;

        fclose(yyin);
        for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
            (*it).second->free();
        };

        if(verbose) {
            cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
			cout<< "disk time " << ( tot_disk / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
        };
    }
    else {
        //context = CreateCudaDevice(0, nullptr, verbose);        
        if(!just_once)
            getline(cin, script);

        while (script != "exit" && script != "EXIT") {

            used_vars.clear();
            yy_scan_string(script.c_str());
            scan_state = 0;
            statement_count = 0;
            clean_queues();
            if(yyparse()) {
                printf("SQL scan parse failed \n");
                getline(cin, script);
                continue;
            };

            scan_state = 1;

            load_vars();

            statement_count = 0;
            clean_queues();
			filter_var.clear();
            yy_scan_string(script.c_str());
            std::clock_t start1 = std::clock();
			curr_time = time(0)*1000;
			
            if(!yyparse()) {
                if(verbose)
                    cout << "SQL scan parse worked " <<  endl;
            };
            for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
                (*it).second->free();
            };
            varNames.clear();

            if(verbose) {
                cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << endl;
            };
            if(!just_once)
                getline(cin, script);
            else
                script = "exit";
        };

        while(!buffer_names.empty()) {
			hipHostFree(buffers[buffer_names.front()]);
            buffer_sizes.erase(buffer_names.front());
            buffers.erase(buffer_names.front());
            buffer_names.pop();
        };
		for(auto it = index_buffers.begin(); it != index_buffers.end();it++) {
			hipHostFree(it->second);
        };
		for(auto it = idx_vals.begin(); it != idx_vals.end();it++) {
			hipFree(it->second);
		idx_vals.clear();	
    };
	

    };
    if(save_dict) {
        save_col_data(data_dict,"data.dictionary");
	};	

    if(alloced_sz) {
        hipFree(alloced_tmp);
        alloced_sz = 0;
    };
	if(scratch.size()) {
		scratch.resize(0);
		scratch.shrink_to_fit();
	};	
	if(rcol_dev.size()) {
		rcol_dev.resize(0);
		rcol_dev.shrink_to_fit();
	};
	if(ranj.size()) {
		ranj.resize(0);
		ranj.shrink_to_fit();
	};	
    return 0;
}



//external c global to report errors
//char gpcACC_err[4048];


int gpcACCExecute(char *s)
{
    YY_BUFFER_STATE bp;

    total_buffer_size = 0;
    scan_state = 0;
    load_col_data(data_dict, "data.dictionary");
    std::clock_t start;

    if(verbose)
        start = std::clock();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    int ret = yyparse();
    //printf("execute: returned [%d]\n", ret);
    if(!ret) {
        if(verbose)
            cout << "SQL scan parse worked" << endl;
    }

    scan_state = 1;
    load_vars();
    statement_count = 0;
    clean_queues();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    if(!yyparse()) {
        if(verbose)
            cout << "SQL scan parse worked " << endl;
    }
    else
        cout << "SQL scan parse failed" << endl;

    yy_delete_buffer(bp);

    // Clear Vars
    for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
        (*it).second->free();
    };
    varNames.clear();

    if(verbose)
        cout<< "statement time " <<  ( ( std::clock() - start ) / (double)CLOCKS_PER_SEC ) << endl;
    if(save_dict)
        save_col_data(data_dict,"data.dictionary");
    return ret;
}


