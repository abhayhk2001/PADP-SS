#include "hip/hip_runtime.h"
#include "merge.h"

using namespace std;

void process_error(int severity, string err);	// this should probably live in a utils header file


#if defined(_MSC_VER)
#define BIG_CONSTANT(x) (x)
// Other compilers
#else   // defined(_MSC_VER)
#define BIG_CONSTANT(x) (x##LLU)
#endif // !defined(_MSC_VER)

unsigned int hash_seed;

struct float_avg
{
	__host__  float_type operator()(const float_type &lhs, const int_type &rhs) const {
		return lhs/rhs;
	}
};

struct float_avg1
{
	__host__  float_type operator()(const int_type &lhs, const int_type &rhs) const {
		return ((float_type)lhs)/rhs;
	}
};

struct div100
{
	__host__  int_type operator()(const int_type &lhs, const int_type &rhs) const {
		return (lhs*100)/rhs;
	}
};

thrust::host_vector<unsigned long long int> h_merge;

using namespace std;
using namespace thrust::placeholders;


void create_c(CudaSet* c, CudaSet* b)
{
	c->not_compressed = 1;
	c->segCount = 1;
	c->columnNames = b->columnNames;
	h_merge.clear();
	c->cols = b->cols;
	c->type = b->type;
	c->decimal = b->decimal;
	c->decimal_zeroes = b->decimal_zeroes;
	c->grp_type = b->grp_type;
	c->ts_cols = b->ts_cols;

	for(unsigned int i=0; i < b->columnNames.size(); i++) {
		if (b->type[b->columnNames[i]] == 0) {
			c->h_columns_int[b->columnNames[i]] = thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >();
			c->d_columns_int[b->columnNames[i]] = thrust::device_vector<int_type>();
			if(b->string_map.find(b->columnNames[i]) != b->string_map.end()) {
				c->string_map[b->columnNames[i]] = b->string_map[b->columnNames[i]];
			};
		}
		else
			if (b->type[b->columnNames[i]] == 1) {
				c->h_columns_float[b->columnNames[i]] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >();
				c->d_columns_float[b->columnNames[i]] = thrust::device_vector<float_type>();
			}
			else {
				c->h_columns_char[b->columnNames[i]] = nullptr;
				c->d_columns_char[b->columnNames[i]] = nullptr;
				c->char_size[b->columnNames[i]] = b->char_size[b->columnNames[i]];
			};
	};
}

void add(CudaSet* c, CudaSet* b, queue<string> op_v3, map<string,string> aliases,
         vector<thrust::device_vector<int_type> >& distinct_tmp, vector<thrust::device_vector<int_type> >& distinct_val,
         vector<thrust::device_vector<int_type> >& distinct_hash, CudaSet* a)
{

	if (c->columnNames.empty()) {
		// create d_columns and h_columns
		create_c(c,b);
	}

	size_t cycle_sz = op_v3.size();

	vector<string> opv;
	for(unsigned int z = 0; z < cycle_sz; z++) {
		if(std::find(b->columnNames.begin(), b->columnNames.end(), aliases[op_v3.front()]) == b->columnNames.end()) { 
			//cout << "Syntax error: alias " << op_v3.front() << endl;
			//exit(0);
			opv.push_back(op_v3.front());
		}
		else
			opv.push_back(aliases[op_v3.front()]);
		op_v3.pop();
	};


	// create hashes of groupby columns
	unsigned long long int* hashes = new unsigned long long int[b->mRecCount];
	unsigned long long int* sum = new unsigned long long int[cycle_sz*b->mRecCount];

	for(unsigned int z = 0; z < cycle_sz; z++) {
		// b->CopyColumnToHost(opv[z]);
		if(b->type[opv[z]] != 1) {  //int or string
			for(int i = 0; i < b->mRecCount; i++) {
				//memcpy(&sum[i*cycle_sz + z], &b->h_columns_int[opv[z]][i], 8);
				sum[i*cycle_sz + z] = b->h_columns_int[opv[z]][i];
				//cout << "CPY to " << i*cycle_sz + z << " " << opv[z] << " " << b->h_columns_int[opv[z]][i] <<   endl;
				//cout << "SET " << sum[i*cycle_sz + z] << endl;
			};
		}
		else {  //float
			for(int i = 0; i < b->mRecCount; i++) {
				memcpy(&sum[i*cycle_sz + z], &b->h_columns_float[opv[z]][i], 8);
			};
		};
	};

	for(int i = 0; i < b->mRecCount; i++) {
		hashes[i] = MurmurHash64A(&sum[i*cycle_sz], 8*cycle_sz, hash_seed);
		//cout << "hash " << hashes[i] << " " << i*cycle_sz << " "  << sum[i*cycle_sz] << " " << sum[i*cycle_sz + 1] << endl;
	};

	delete [] sum;
	thrust::device_vector<unsigned long long int> d_hashes(b->mRecCount);
	thrust::device_vector<unsigned int> v(b->mRecCount);
	thrust::sequence(v.begin(), v.end(), 0, 1);
	thrust::copy(hashes, hashes+b->mRecCount, d_hashes.begin());

	// sort the results by hash
	thrust::sort_by_key(d_hashes.begin(), d_hashes.end(), v.begin());

	void* d_tmp;
	CUDA_SAFE_CALL(hipMalloc((void **) &d_tmp, b->mRecCount*int_size));

	for(unsigned int i = 0; i < b->columnNames.size(); i++) {

		if(b->type[b->columnNames[i]] == 0 || b->type[b->columnNames[i]] == 2) {
			thrust::device_ptr<int_type> d_tmp_int((int_type*)d_tmp);
			thrust::gather(v.begin(), v.end(), b->d_columns_int[b->columnNames[i]].begin(), d_tmp_int);
			thrust::copy(d_tmp_int, d_tmp_int + b->mRecCount, b->h_columns_int[b->columnNames[i]].begin());
		}
		else
			if(b->type[b->columnNames[i]] == 1) {
				thrust::device_ptr<float_type> d_tmp_float((float_type*)d_tmp);
				thrust::gather(v.begin(), v.end(), b->d_columns_float[b->columnNames[i]].begin(), d_tmp_float);
				thrust::copy(d_tmp_float, d_tmp_float + b->mRecCount, b->h_columns_float[b->columnNames[i]].begin());
			}
	};
	hipFree(d_tmp);

	thrust::host_vector<unsigned long long int> hh = d_hashes;
	char* tmp = new char[max_char(b)*(c->mRecCount + b->mRecCount)];
	c->resize(b->mRecCount);

	//lets merge every column

	for(unsigned int i = 0; i < b->columnNames.size(); i++) {

		if(b->type[b->columnNames[i]] != 1) {

			thrust::merge_by_key(h_merge.begin(), h_merge.end(),
			                     hh.begin(), hh.end(),
			                     c->h_columns_int[c->columnNames[i]].begin(), b->h_columns_int[b->columnNames[i]].begin(),
			                     thrust::make_discard_iterator(), (int_type*)tmp);
			memcpy(thrust::raw_pointer_cast(c->h_columns_int[c->columnNames[i]].data()), (int_type*)tmp, (h_merge.size() + b->mRecCount)*int_size);
		}
		else {
			thrust::merge_by_key(h_merge.begin(), h_merge.end(),
			                     hh.begin(), hh.end(),
			                     c->h_columns_float[c->columnNames[i]].begin(), b->h_columns_float[b->columnNames[i]].begin(),
			                     thrust::make_discard_iterator(), (float_type*)tmp);
			memcpy(thrust::raw_pointer_cast(c->h_columns_float[c->columnNames[i]].data()), (float_type*)tmp, (h_merge.size() + b->mRecCount)*float_size);
		}
	};


	//merge the keys
	thrust::merge(h_merge.begin(), h_merge.end(),
	              hh.begin(), hh.end(), (unsigned long long int*)tmp);

	size_t cpy_sz = h_merge.size() + b->mRecCount;
	h_merge.resize(h_merge.size() + b->mRecCount);
	thrust::copy((unsigned long long int*)tmp, (unsigned long long int*)tmp + cpy_sz, h_merge.begin());

	delete [] tmp;
	delete [] hashes;

}

void count_avg(CudaSet* c,  vector<thrust::device_vector<int_type> >& distinct_hash)
{
	string countstr;
	thrust::equal_to<unsigned long long int> binary_pred;
	thrust::maximum<unsigned long long int> binary_op_max;
	thrust::minimum<unsigned long long int> binary_op_min;

	for(unsigned int i = 0; i < c->columnNames.size(); i++) {
		if(c->grp_type[c->columnNames[i]] == 0) { // COUNT
			countstr = c->columnNames[i];
			break;
		};
	};


	thrust::host_vector<bool> grp;
	size_t res_count;

	if(h_merge.size()) {
		grp.resize(h_merge.size());
		thrust::adjacent_difference(h_merge.begin(), h_merge.end(), grp.begin());
		res_count = h_merge.size() - thrust::count(grp.begin(), grp.end(), 0);
	};


	if (c->mRecCount != 0) {

		//unsigned int dis_count = 0;
		if (h_merge.size()) {
			int_type* tmp =  new int_type[res_count];
			for(unsigned int k = 0; k < c->columnNames.size(); k++)	{

				if(c->grp_type[c->columnNames[k]] <= 2) { //sum || avg || count
					if (c->type[c->columnNames[k]] == 0) { // int
						// check for overflow
						// convert to double, reduce, check if larger than max 64 bit int

						float_type* tmp1 =  new float_type[c->mRecCount];
						float_type* tmp_res = new float_type[res_count];

						for(int z = 0; z < c->mRecCount ; z++)
							tmp1[z] = (float_type)(c->h_columns_int[c->columnNames[k]][z]);

						thrust::reduce_by_key(h_merge.begin(), h_merge.end(), tmp1,
						                      thrust::make_discard_iterator(), tmp_res);

						double max_overflow = 0;
						for(int z = 0; z < res_count; z++) {
							if (tmp_res[z] > 9223372036854775807.0) {
								if(tmp_res[z] - 9223372036854775807.0 > max_overflow)
									max_overflow = tmp_res[z];
							};
						};
						if(max_overflow) {
							unsigned pw = ceil(log10(max_overflow/9223372036854775807.0));
							thrust::transform(c->h_columns_int[c->columnNames[k]].begin(), c->h_columns_int[c->columnNames[k]].end(), thrust::make_constant_iterator((int_type)pow(10, pw)), c->h_columns_int[c->columnNames[k]].begin(), thrust::divides<int_type>());
							c->decimal_zeroes[c->columnNames[k]] = c->decimal_zeroes[c->columnNames[k]] - pw;
						};

						delete [] tmp1;
						delete [] tmp_res;

						thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
						                      thrust::make_discard_iterator(), tmp);
						c->h_columns_int[c->columnNames[k]].resize(res_count);
						thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
					}
					else
						if (c->type[c->columnNames[k]] == 1 ) { // float
							float_type* tmp1 =  new float_type[res_count];
							thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
							                      thrust::make_discard_iterator(), tmp1);
							c->h_columns_float[c->columnNames[k]].resize(res_count);
							thrust::copy(tmp1, tmp1 + res_count, c->h_columns_float[c->columnNames[k]].begin());
							delete [] tmp1;
						};
				}
				if(c->grp_type[c->columnNames[k]] == 4) { //min
					if (c->type[c->columnNames[k]] == 0 ) { // int
						thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
						                      thrust::make_discard_iterator(), tmp, binary_pred, binary_op_min);
						c->h_columns_int[c->columnNames[k]].resize(res_count);
						thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
					}
					else
						if (c->type[c->columnNames[k]] == 1 ) { // float
							c->h_columns_float[c->columnNames[k]].resize(res_count);
							thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
							                      thrust::make_discard_iterator(), c->h_columns_float[c->columnNames[k]].begin(), binary_pred, binary_op_min);
						};
				}
				if(c->grp_type[c->columnNames[k]] == 5) { //max
					if (c->type[c->columnNames[k]] == 0 ) { // int
						int_type* tmp =  new int_type[res_count];
						thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
						                      thrust::make_discard_iterator(), tmp, binary_pred, binary_op_max);
						c->h_columns_int[c->columnNames[k]].resize(res_count);
						thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
						delete [] tmp;
					}
					else
						if (c->type[c->columnNames[k]] == 1 ) { // float
							c->h_columns_float[c->columnNames[k]].resize(res_count);
							thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
							                      thrust::make_discard_iterator(), c->h_columns_float[c->columnNames[k]].begin(), binary_pred, binary_op_max);
						};
				}
				else
					if(c->grp_type[c->columnNames[k]] == 3) { //no group function
						if (c->type[c->columnNames[k]] == 0 || c->type[c->columnNames[k]] == 2) { // int
							thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
							                      thrust::make_discard_iterator(), tmp, binary_pred, binary_op_max);
							c->h_columns_int[c->columnNames[k]].resize(res_count);
							thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
						}
						else
							if (c->type[c->columnNames[k]] == 1 ) { // float
								c->h_columns_float[c->columnNames[k]].resize(res_count);
								thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
								                      thrust::make_discard_iterator(), c->h_columns_float[c->columnNames[k]].begin(), binary_pred, binary_op_max);
							}
					};
			};
			c->mRecCount = res_count;
			delete [] tmp;
		};

		for(unsigned int k = 0; k < c->columnNames.size(); k++)	{
			if(c->grp_type[c->columnNames[k]] == 1) {   // AVG

				if (c->type[c->columnNames[k]] == 0 ) { // int

					if(c->decimal_zeroes[c->columnNames[k]] <= 2) {
						thrust::transform(c->h_columns_int[c->columnNames[k]].begin(), c->h_columns_int[c->columnNames[k]].begin() + c->mRecCount,
						                  c->h_columns_int[countstr].begin(), c->h_columns_int[c->columnNames[k]].begin(), div100());
						c->decimal_zeroes[c->columnNames[k]] = c->decimal_zeroes[c->columnNames[k]] + 2;
					}
					else {
						thrust::transform(c->h_columns_int[c->columnNames[k]].begin(), c->h_columns_int[c->columnNames[k]].begin() + c->mRecCount,
						                  c->h_columns_int[countstr].begin(), c->h_columns_int[c->columnNames[k]].begin(), thrust::divides<int_type>());
					};
					c->grp_type[c->columnNames[k]] = 3;
				}
				else {              // float
					thrust::transform(c->h_columns_float[c->columnNames[k]].begin(), c->h_columns_float[c->columnNames[k]].begin() + c->mRecCount,
					                  c->h_columns_int[countstr].begin(), c->h_columns_float[c->columnNames[k]].begin(), float_avg());
				};
			}
			else
				if(c->grp_type[c->columnNames[k]] == 6) {
				}
				else
					if(c->grp_type[c->columnNames[k]] == 2) {

					};
		};

	};

	c->segCount = 1;
	c->maxRecs = c->mRecCount;
};

